#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/stat.h>
#include <dirent.h>
//#include <png.h>
//#include <jpeglib.h>
#include <omp.h>
#include "ImageIO/image.h"
#include "sbr.h"
#include "sbr_opt.h"
#include "water.h"

__device__ static float atomicMaxFloat(float* address, float val){
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed, __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

/*
if((threadIdx.x==0)&&(threadIdx.y==0)){
    printf("AAAAAAAAAAAAAAAAAA\n");
    for(int j=0;j<stroke_length_max;j++){
        for(int i=0;i<stroke_length_max;i++){
            printf("%3d ", s_test_Canvas_B[i+j*stroke_length_max]);
        }
        printf("\n");
    }
}
__syncthreads();
*/

//ストロークを沢山描いて各ストロークの改善値を計算する関数
__global__ void gpu_calculate_best_stroke(int *dev_GLOBAL_improved_value_map, float *dev_PerlinNoise, int *dev_cmprR,	int *dev_cmprG,	int *dev_cmprB,	int *dev_nimgR, int *dev_nimgG,	int *dev_nimgB,
                                int *dev_best_stroke_map_pnum, float *dev_best_stroke_map_point_x, float *dev_best_stroke_map_point_y, int *dev_best_stroke_map_R, int *dev_best_stroke_map_G,
                                int *dev_best_stroke_map_B, float *dev_sobel_abs, float *dev_sobel_angle, float *dev_grad_hx, float *dev_grad_hy, float *dev_in_Lab_L, float *dev_in_Lab_a,
                                float *dev_in_Lab_b, char *dev_M, float *dev_u, float *dev_new_u, float *dev_v, float *dev_new_v, float *dev_p, float *dev_gR, float *dev_gG, float *dev_gB,
                                float *dev_dR, float *dev_dG, float *dev_dB, float *dev_new_gR,	float *dev_new_gG, float *dev_new_gB, float *dev_new_dR, float *dev_new_dG, float *dev_new_dB,
                                float *dev_gauss_filter, float *dev_gauss_M, float *dev_h, int width, int height, int t){

    int global_blockID = blockIdx.x + blockIdx.y * gridDim.x;
    int stroke_length_max = opt_thick_max*(opt_max_stroke+2);
    int SubImageIndex = global_blockID * stroke_length_max * stroke_length_max;
    int SubImageIndex_uv = global_blockID * stroke_length_max * (stroke_length_max+1);

    int x,y,i,j,k,l,yc,xc,sy,sx,ly,lx;
    int pnum, peak, stroke_partition, left_end, right_end, upper_end, lower_end, stroke_length_x, stroke_length_y;
    float temp_R, temp_G, temp_B, Can_L, Can_a, Can_b, testCan_L, testCan_a, testCan_b, Lab_x, Lab_y, Lab_z, error_before, error_after;
    float format_theta, scale, temp_x, temp_y, tmpSP_start_x, tmpSP_start_y, tmpSP_end_x, tmpSP_end_y, tmpSP0_x, tmpSP0_y, tmpSP1_x, tmpSP1_y;
    float UV_var_t, A, B, p_grad, delta, sum, filter_sum, down, up, down_ratio, up_ratio, pigment_density;
    
    int tmp_density_R = 255 - 255 * opt_ratio;//描画色の濃度
    int tmp_density_G = 255 - 255 * opt_ratio;//描画色の濃度
    int tmp_density_B = 255 - 255 * opt_ratio;//描画色の濃度

    int w = (int)(ceil(3.0*opt_K/6.0+0.5)*2-1); //ガウシアンフィルタ用
    int c = (w-1)/2;                            //ガウシアンフィルタ用

	__shared__ int s_improved_value[1];
	__shared__ float s_diff_sum[1];
	__shared__ int s_offscrn_count[1];
    __shared__ float s_theta[1];
    __shared__ float s_histogram[opt_histogram_partition];
    __shared__ float s_error_sum[1];
    __shared__ short s_test_Canvas_R[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ short s_test_Canvas_G[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ short s_test_Canvas_B[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ float s_max_velocity[1];
    __shared__ float s_delta_MAX[1];

    for(y=blockIdx.y; y<height; y=y+gridDim.y) {
		for(x=blockIdx.x; x<width; x=x+gridDim.x) {

			// 改善値が計算済みならSkip
			if(dev_GLOBAL_improved_value_map[x+y*width] != UNCALCULATED) continue;

			pnum = peak = 0;
            if((threadIdx.x==0)&&(threadIdx.y==0)){
                s_diff_sum[0] = 0;
                s_offscrn_count[0] = 0;
            }
            __syncthreads();

			//ウィンドウの中の差分の合計を並列に計算
			for(yc=threadIdx.y-t; yc<=t; yc=yc+blockDim.y) {
				for(xc=threadIdx.x-t; xc<=t; xc=xc+blockDim.x) {
					if((x+xc)<0 || (x+xc)>(width-1) || (y+yc)<0 || (y+yc)>(height-1)){
                        atomicAdd(&s_offscrn_count[0], 1);
					}else{

                        //描画色の平均を求める用
						atomicAdd(&dev_best_stroke_map_R[x+y*width], dev_cmprR[(x+xc)+(y+yc)*width]);
						atomicAdd(&dev_best_stroke_map_G[x+y*width], dev_cmprG[(x+xc)+(y+yc)*width]);
						atomicAdd(&dev_best_stroke_map_B[x+y*width], dev_cmprB[(x+xc)+(y+yc)*width]);

                        //誤差の平均を求める用
                        atomicAdd(&s_diff_sum[0], fabsf((float)(dev_nimgR[(x+xc)+(y+yc)*width] - dev_cmprR[(x+xc)+(y+yc)*width])));
						atomicAdd(&s_diff_sum[0], fabsf((float)(dev_nimgG[(x+xc)+(y+yc)*width] - dev_cmprG[(x+xc)+(y+yc)*width])));
						atomicAdd(&s_diff_sum[0], fabsf((float)(dev_nimgB[(x+xc)+(y+yc)*width] - dev_cmprB[(x+xc)+(y+yc)*width])));
					}
				}
			}
            __syncthreads();

			//1つの代表スレッドを起動
			if((threadIdx.x==0)&&(threadIdx.y==0)){

                //描画色の平均を計算
				dev_best_stroke_map_R[x+y*width] = dev_best_stroke_map_R[x+y*width] / ((2*t+1)*(2*t+1)-s_offscrn_count[0]);
				dev_best_stroke_map_G[x+y*width] = dev_best_stroke_map_G[x+y*width] / ((2*t+1)*(2*t+1)-s_offscrn_count[0]);
				dev_best_stroke_map_B[x+y*width] = dev_best_stroke_map_B[x+y*width] / ((2*t+1)*(2*t+1)-s_offscrn_count[0]);

                //誤差の平均を計算
                s_diff_sum[0] = s_diff_sum[0] / ((2*t+1)*(2*t+1)-s_offscrn_count[0]) / 3;
            }
            __syncthreads();

            //差分の合計平均(画素当たりの差分)が一定以上ならストローク開始位置とする
            if(s_diff_sum[0] < opt_window_diff_border) {
                dev_GLOBAL_improved_value_map[x+y*width] = SMALL_DIFF;
                continue;
            }

            //1つの代表スレッドが第一制御点の座標を格納
			if((threadIdx.x==0)&&(threadIdx.y==0)){
                dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = x+0.5;
				dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = y+0.5;
			}
			__syncthreads();

			pnum = 1;		//第1制御点確定

            //SharedMemory初期化
            for(i=threadIdx.x+threadIdx.y*blockDim.x; i<opt_histogram_partition ; i+= blockDim.x*blockDim.y) {s_histogram[i]=0;}
            __syncthreads();

            //sobelからヒストグラムを作成
            for(sy=threadIdx.y+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]-t-1; sy<=dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]+t+1; sy+=blockDim.y) {
                for(sx=threadIdx.x+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]-t-1; sx<=dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]+t+1; sx+=blockDim.x) {
                    if(!(sx<1 || sx>width-2 || sy<1 || sy>height-2)){
                        atomicAdd(&s_histogram[(int)((dev_sobel_angle[sx+sy*width]/PI)*opt_histogram_partition)], dev_sobel_abs[sx+sy*width]);
                    }
                }
            }
            __syncthreads();

            //1つの代表スレッドを起動
            if((threadIdx.x==0)&&(threadIdx.y==0)){

                //ヒストグラムの中で最も大きい値を探索
                for(i=0; i<opt_histogram_partition; i++){
                    if(s_histogram[peak] < s_histogram[i]) peak = i;
                }

                //thetaを計算
                s_theta[0] = ((float)peak/opt_histogram_partition)*PI+((PI/opt_histogram_partition)/2);

                //次の制御点を方向から計算
                dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = t*cos(s_theta[0])+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
                dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = t*sin(s_theta[0])+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];
                if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] < 0)
                    dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = 0;
                if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] < 0)
                    dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = 0;
                if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] >= width)
                    dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = width-1;
                if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] >= height)
                    dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = height-1;
            }
            __syncthreads();

            //二つ目の描画点周りの色が描画色と一致するか確認する
            error_before = error_after = 0;
            if((threadIdx.x==0)&&(threadIdx.y==0)){
                s_offscrn_count[0] = 0;
                s_error_sum[0] = 0;
            }
            __syncthreads();

            for(ly=threadIdx.y+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]-t; ly<=dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]+t; ly+=blockDim.y) {
                for(lx=threadIdx.x+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]-t; lx<=dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]+t; lx+=blockDim.x) {
                    if(lx<0 || lx>width-1 || ly<0 || ly>height-1) {
                        atomicAdd(&s_offscrn_count[0], 1);
                    }else if( (lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])*(lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])+(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum])*(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]) > t*t ){
                        atomicAdd(&s_offscrn_count[0], 1);
                    }else{

                        //塗る前のLabを計算
                        temp_R = dev_nimgR[lx+ly*width];
                        temp_G = dev_nimgG[lx+ly*width];
                        temp_B = dev_nimgB[lx+ly*width];
                        temp_R = temp_R / 255.0;
                        temp_G = temp_R / 255.0;
                        temp_B = temp_R / 255.0;
                        temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                        temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                        temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                        Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                        Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                        Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                        Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                        Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                        Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                        Can_L = (116 * Lab_y) - 16;
                        Can_a = 500 * (Lab_x - Lab_y);
                        Can_b = 200 * (Lab_y - Lab_z);

                        //塗った後のLabを計算
                        temp_R = dev_nimgR[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_R[x+y*width] * opt_ratio;
                        temp_G = dev_nimgG[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_G[x+y*width] * opt_ratio;
                        temp_B = dev_nimgB[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_B[x+y*width] * opt_ratio;
                        temp_R = temp_R / 255.0;
                        temp_G = temp_R / 255.0;
                        temp_B = temp_R / 255.0;
                        temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                        temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                        temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                        Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                        Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                        Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                        Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                        Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                        Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                        testCan_L = (116 * Lab_y) - 16;
                        testCan_a = 500 * (Lab_x - Lab_y);
                        testCan_b = 200 * (Lab_y - Lab_z);

                        //描画前後のLab空間におけるユークリッド距離誤算の変化を計算
                        error_before = sqrt( powf(dev_in_Lab_L[lx+ly*width]-Can_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-Can_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-Can_b, 2) );
                        error_after = sqrt( powf(dev_in_Lab_L[lx+ly*width]-testCan_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-testCan_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-testCan_b, 2) );
                        atomicAdd(&s_error_sum[0], error_before-error_after);
                    }
                }
            }
            __syncthreads();

            //二つ目の制御点周りの色が描画色としきい値以上の差を持つなら描画せず反対方向の制御点を見る
			if(s_error_sum[0] < opt_color_diff_border){

                //1つの代表スレッドを起動してもう1つの勾配垂直の点を代入
                if((threadIdx.x==0)&&(threadIdx.y==0)){
                    s_theta[0] += PI;
                    dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = t*cos(s_theta[0])+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
                    dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = t*sin(s_theta[0])+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];
                    if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] < 0)
                        dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = 0;
                    if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] < 0)
                        dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = 0;
                    if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] >= width)
                        dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = width-1;
                    if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] >= height)
                        dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = height-1;
                }
                __syncthreads();

                //反対方向の第2点の描画点周りの色が描画色と一致するか確認する
                error_before = error_after = 0;
                if((threadIdx.x==0)&&(threadIdx.y==0)){
                    s_offscrn_count[0] = 0;
                    s_error_sum[0] = 0;
                }
                __syncthreads();

                for(ly=threadIdx.y+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]-t; ly<=dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]+t; ly+=blockDim.y) {
                    for(lx=threadIdx.x+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]-t; lx<=dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]+t; lx+=blockDim.x) {
                        if(lx<0 || lx>width-1 || ly<0 || ly>height-1) {
                            atomicAdd(&s_offscrn_count[0], 1);
                        }else if( (lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])*(lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])+(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum])*(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]) > t*t ){
                            atomicAdd(&s_offscrn_count[0], 1);
                        }else{

                            //塗る前のLabを計算
                            temp_R = dev_nimgR[lx+ly*width];
                            temp_G = dev_nimgG[lx+ly*width];
                            temp_B = dev_nimgB[lx+ly*width];
                            temp_R = temp_R / 255.0;
                            temp_G = temp_R / 255.0;
                            temp_B = temp_R / 255.0;
                            temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                            temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                            temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                            Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                            Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                            Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                            Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                            Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                            Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                            Can_L = (116 * Lab_y) - 16;
                            Can_a = 500 * (Lab_x - Lab_y);
                            Can_b = 200 * (Lab_y - Lab_z);

                            //塗った後のLabを計算
                            temp_R = dev_nimgR[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_R[x+y*width] * opt_ratio;
                            temp_G = dev_nimgG[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_G[x+y*width] * opt_ratio;
                            temp_B = dev_nimgB[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_B[x+y*width] * opt_ratio;
                            temp_R = temp_R / 255.0;
                            temp_G = temp_R / 255.0;
                            temp_B = temp_R / 255.0;
                            temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                            temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                            temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                            Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                            Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                            Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                            Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                            Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                            Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                            testCan_L = (116 * Lab_y) - 16;
                            testCan_a = 500 * (Lab_x - Lab_y);
                            testCan_b = 200 * (Lab_y - Lab_z);

                            //描画前後のLab空間におけるユークリッド距離誤算の変化を計算
                            error_before = sqrt( powf(dev_in_Lab_L[lx+ly*width]-Can_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-Can_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-Can_b, 2) );
                            error_after = sqrt( powf(dev_in_Lab_L[lx+ly*width]-testCan_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-testCan_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-testCan_b, 2) );
                            atomicAdd(&s_error_sum[0], error_before-error_after);
                        }
                    }
                }
                __syncthreads();

                //どちらの第2点も不適切なら描画をせず次のループへ
                if(s_error_sum[0] < opt_color_diff_border){
                    dev_GLOBAL_improved_value_map[x+y*width] = MIN_STROKE;
                    continue;
                }
            }

            pnum=2;		//第2制御点確定

            //ストロークが最大opt_max_strokeになるまでは伸ばし続ける
            while(pnum < opt_max_stroke){

                //thetaの値を保存
                format_theta = s_theta[0];

                //第pnum点周りにおいて、sobelからヒストグラムを作成
                peak = 0;//初期化
                for(i=threadIdx.x+threadIdx.y*blockDim.x; i<opt_histogram_partition ; i+= blockDim.x*blockDim.y) {s_histogram[i]=0;}//ヒストグラムを初期化
                __syncthreads();

                for(sy=threadIdx.y+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]-t-1; sy<=dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]+t+1; sy=sy+blockDim.y) {
                    for(sx=threadIdx.x+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]-t-1; sx<=dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]+t+1; sx=sx+blockDim.x) {
                        if(!(sx<1 || sx>width-2 || sy<1 || sy>height-2))
                            atomicAdd(&s_histogram[(int)((dev_sobel_angle[sx+sy*width]/PI)*opt_histogram_partition)], dev_sobel_abs[sx+sy*width]);
                    }
                }
                __syncthreads();

                //1つの代表スレッドを起動
                if((threadIdx.x==0)&&(threadIdx.y==0)){

                    //ヒストグラムの中で最も大きい値を探索
                    for(i=0; i<opt_histogram_partition; i++){
                        if(s_histogram[peak] < s_histogram[i]) peak = i;
                    }

                    //thetaを計算
                    s_theta[0] = ((float)peak/opt_histogram_partition)*PI+((PI/opt_histogram_partition)/2);

                    //制御点の為す角が急峻になるようなら逆方向に角度を取る
					if( (s_theta[0] < format_theta-PI/2) || (s_theta[0] > format_theta+PI/2) ) {s_theta[0] += PI;}

                    //次の制御点を方向から計算
                    dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = t*cos(s_theta[0])+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
                    dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = t*sin(s_theta[0])+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];
                    if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] < 0)
                        dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = 0;
                    if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] < 0)
                        dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = 0;
                    if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] >= width)
                        dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum] = width-1;
                    if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] >= height)
                        dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum] = height-1;
                }
                __syncthreads();

                //pnum+1目の描画点周りの色が描画色と一致するか確認する
                error_before = error_after = 0;
                if((threadIdx.x==0)&&(threadIdx.y==0)){
                    s_offscrn_count[0] = 0;
                    s_error_sum[0] = 0;
                }
                __syncthreads();

                for(ly=threadIdx.y+dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]-t; ly<=dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]+t; ly=ly+blockDim.y) {
                    for(lx=threadIdx.x+dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]-t; lx<=dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum]+t; lx=lx+blockDim.x) {
                        if(lx<0 || lx>width-1 || ly<0 || ly>height-1) {
                            atomicAdd(&s_offscrn_count[0], 1);
                        }else if( (lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])*(lx-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum])+(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum])*(ly-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum]) > t*t ){
                            atomicAdd(&s_offscrn_count[0], 1);
                        }else{

                            //塗る前のLabを計算
                            temp_R = dev_nimgR[lx+ly*width];
                            temp_G = dev_nimgG[lx+ly*width];
                            temp_B = dev_nimgB[lx+ly*width];
                            temp_R = temp_R / 255.0;
                            temp_G = temp_R / 255.0;
                            temp_B = temp_R / 255.0;
                            temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                            temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                            temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                            Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                            Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                            Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                            Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                            Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                            Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                            Can_L = (116 * Lab_y) - 16;
                            Can_a = 500 * (Lab_x - Lab_y);
                            Can_b = 200 * (Lab_y - Lab_z);

                            //塗った後のLabを計算
                            temp_R = dev_nimgR[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_R[x+y*width] * opt_ratio;
                            temp_G = dev_nimgG[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_G[x+y*width] * opt_ratio;
                            temp_B = dev_nimgB[lx+ly*width] * (1-opt_ratio) + dev_best_stroke_map_B[x+y*width] * opt_ratio;
                            temp_R = temp_R / 255.0;
                            temp_G = temp_R / 255.0;
                            temp_B = temp_R / 255.0;
                            temp_R = temp_R > 0.04045 ? powf(((temp_R + 0.055) / 1.055), 2.4) : (temp_R / 12.92);
                            temp_G = temp_G > 0.04045 ? powf(((temp_G + 0.055) / 1.055), 2.4) : (temp_G / 12.92);
                            temp_B = temp_B > 0.04045 ? powf(((temp_B + 0.055) / 1.055), 2.4) : (temp_B / 12.92);
                            Lab_x = ((temp_R * 0.4124) + (temp_G * 0.3576) + (temp_B * 0.1805)) * 100 / 95.047;
                            Lab_y = (temp_R * 0.2126) + (temp_G * 0.7152) + (temp_B * 0.0722);
                            Lab_z = ((temp_R * 0.0193) + (temp_G * 0.1192) + (temp_B * 0.9505)) * 100 / 108.883;
                            Lab_x = Lab_x > 0.008856 ? powf(Lab_x, 1 / 3.0) : (7.787 * Lab_x) + (4 / 29.0);
                            Lab_y = Lab_y > 0.008856 ? powf(Lab_y, 1 / 3.0) : (7.787 * Lab_y) + (4 / 29.0);
                            Lab_z = Lab_z > 0.008856 ? powf(Lab_z, 1 / 3.0) : (7.787 * Lab_z) + (4 / 29.0);
                            testCan_L = (116 * Lab_y) - 16;
                            testCan_a = 500 * (Lab_x - Lab_y);
                            testCan_b = 200 * (Lab_y - Lab_z);

                            //描画前後のLab空間におけるユークリッド距離誤算の変化を計算
                            error_before = sqrt( powf(dev_in_Lab_L[lx+ly*width]-Can_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-Can_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-Can_b, 2) );
                            error_after = sqrt( powf(dev_in_Lab_L[lx+ly*width]-testCan_L, 2) + powf(dev_in_Lab_a[lx+ly*width]-testCan_a, 2) + powf(dev_in_Lab_b[lx+ly*width]-testCan_b, 2) );
                            atomicAdd(&s_error_sum[0], error_before-error_after);
                        }
                    }
                }
                __syncthreads();

                if(s_error_sum[0] < opt_color_diff_border) break;
                else pnum++;
            }

            //1つの代表スレッドが制御点の個数を記録
            if((threadIdx.x==0)&&(threadIdx.y==0)){
                dev_best_stroke_map_pnum[x+y*width] = pnum;
            }
            __syncthreads();
            
            //////////ストローク形状の計算終了//////////

            //if((threadIdx.x==0)&&(threadIdx.y==0))    printf("(%d,%d)Stroke_form_end\n",x,y);
            //__syncthreads();

            //制御点の個数が足りない場合
            if(pnum < opt_min_stroke){
                dev_GLOBAL_improved_value_map[x+y*width] = MIN_STROKE;
                continue;
            }

            //////////Paint_Water_Stroke(試しに描いてみて誤差を確認)//////////

            //ストローク点を囲む端の座標を特定
            left_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0];       //切り捨て
            right_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]+1;    //切り上げ
            upper_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0];      //切り捨て
            lower_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]+1;    //切り上げ
            for(i=1; i<pnum; i++){
                if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i] < left_end) left_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i];
                if(right_end < dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]) right_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]+1;
                if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i] < upper_end) upper_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i];
                if(lower_end < dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]) lower_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]+1;
            }

            //ストローク半径分、端座標を膨張
            left_end-=t; right_end+=t; upper_end-=t; lower_end+=t;
            if(left_end < 0) left_end = 0;
            if(width < right_end) right_end = width-1;
            if(upper_end < 0) upper_end=0;
            if(height < lower_end) lower_end = height-1;

            stroke_length_x = right_end - left_end; //ストロークの横の長さ
            stroke_length_y = lower_end - upper_end; //ストロークの縦の長さ

            //各パラメータの初期化
            for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                    dev_M[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_p[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_gR[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_gG[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_gB[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_gR[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_gG[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_gB[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_dR[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_dG[SubImageIndex+j+i*stroke_length_max] = 0;
                    dev_new_dB[SubImageIndex+j+i*stroke_length_max] = 0;
                }
            }

            //uとvを初期化
            for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                    dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = 0;
                    dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = 0;
                }
            }
            for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                    dev_v[SubImageIndex_uv+j+i*stroke_length_max] = 0;
                    dev_new_v[SubImageIndex_uv+j+i*stroke_length_max] = 0;
                }
            }

            //影響箇所をコピー
            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    dev_dR[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgR[(left_end+i)+(upper_end+j)*width] / 255.0;
                    dev_dG[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgG[(left_end+i)+(upper_end+j)*width] / 255.0;
                    dev_dB[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgB[(left_end+i)+(upper_end+j)*width] / 255.0;
                    s_test_Canvas_R[i+j*stroke_length_max] = dev_nimgR[(left_end+i)+(upper_end+j)*width];
                    s_test_Canvas_G[i+j*stroke_length_max] = dev_nimgG[(left_end+i)+(upper_end+j)*width];
                    s_test_Canvas_B[i+j*stroke_length_max] = dev_nimgB[(left_end+i)+(upper_end+j)*width];
                    dev_h[SubImageIndex+i+j*stroke_length_max] = dev_PerlinNoise[(left_end+i)+(upper_end+j)*width];
                }
            }
            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] = (-1)*dev_grad_hx[(left_end+i)+(upper_end+j)*(width+1)];
                }
            }
            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    dev_v[SubImageIndex_uv+i+j*stroke_length_max] = (-1)*dev_grad_hy[(left_end+i)+(upper_end+j)*width];
                }
            }
            __syncthreads();

            //////////set_WetStroke(ストローク点に従いウェットエリアと水量を計算)//////////
            if(pnum == 2){//もし制御点が2つの時は直線を引く
                stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1])
                                    + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]); //線分の分割数                    
                
                for(k=0; k<=stroke_partition; k++){
                    scale = (float)k / stroke_partition;
                    temp_x = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] + (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0])*scale;
                    temp_y = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] + (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0])*scale;

                    for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                        for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                            if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                                if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                                    dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                                    dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                                    dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                                    dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                                    dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                                }
                            }
                        }
                    }
                }
            
            //制御点が3つ以上の場合
            }else{

                //1.最初の制御点について
                //端の一つ外の制御点を適当に決める
                tmpSP_start_x = 2*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
                tmpSP_start_y = 2*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];
                
                //ベジエ曲線用の中間点を2点決める
                tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1]-tmpSP_start_x)/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0];
                tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]-tmpSP_start_y)/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0];
                tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+2])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
                tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+2])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];

                //線分の分割数
                stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1])
                                + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]);
                
                for(k=0; k<=stroke_partition; k++){
                    scale = (float)k / stroke_partition;
                    temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
                    temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];

                    for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                        for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                            if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                                if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                                    dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                                    dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                                    dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                                    dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                                    dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                                }
                            }
                        }
                    }
                }
                __syncthreads();

                //2.中間の制御点について
                for(i=1; i<pnum-2; i++){

                    //ベジエ曲線用の中間点を2点決める
                    tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i-1])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i];
                    tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i-1])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i];
                    tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+2])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1];
                    tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+2])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1];

                    //線分の分割数
                    stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1])
                                    + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1]);

                    for(j=0; j<=stroke_partition; j++){
                        scale = (float)j / stroke_partition;
                        temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1];
                        temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1];

                        for(k=threadIdx.y+temp_y-t-upper_end; k<=temp_y+t-upper_end; k+=blockDim.y){
                            for(l=threadIdx.x+temp_x-t-left_end; l<=temp_x+t-left_end; l+=blockDim.x){
                                if((l>=0) && (l<=stroke_length_x) && (k>=0) && (k<=stroke_length_y)){//画像内かどうか
                                    if((l-temp_x+left_end)*(l-temp_x+left_end)+(k-temp_y+upper_end)*(k-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                                        dev_M[SubImageIndex+l+k*stroke_length_max] = 1;
                                        dev_gR[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                                        dev_gG[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                                        dev_gB[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                                        dev_p[SubImageIndex+l+k*stroke_length_max] = 1.0;
                                    }
                                }
                            }
                        }
                    }
                }

                //3.最後の制御点について
                //端の一つ外の制御点を適当に決める
                tmpSP_end_x = 2*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2];
                tmpSP_end_y = 2*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2];

                //ベジエ曲線用の中間点を2点決める
                tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-3])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2];
                tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-3])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2];
                tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2]-tmpSP_end_x)/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
                tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2]-tmpSP_end_y)/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];

                //線分の分割数
                stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1])
                                + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]);

                for(k=0; k<=stroke_partition; k++){
                    scale = (float)k / stroke_partition;
                    temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
                    temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];

                    for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                        for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                            if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                                if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                                    dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                                    dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                                    dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                                    dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                                    dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();

            //if((threadIdx.x==0)&&(threadIdx.y==0))    printf("(%d,%d)Set_WetStroke_end\n",x,y);
            //__syncthreads();

            //時間の経過を表すループ
            for (float time=0; time<opt_SoakTime; time+=opt_SoakTimeStep){

                //////////UpdateVelocities(一定時間経過後の速度変化を計算)//////////
                if((threadIdx.x==0)&&(threadIdx.y==0)) s_max_velocity[0] = 0;//最大初速度を初期化
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max]==1){
                            atomicMaxFloat(&s_max_velocity[0], fabsf(dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]));
                            atomicMaxFloat(&s_max_velocity[0], fabsf(dev_v[SubImageIndex_uv+i+j*stroke_length_max]));
                        }
                    }
                }
                __syncthreads();

                if(s_max_velocity[0] > opt_StopSoakVero) break; //水速度が暴走したときは停止
                UV_var_t = fminf(opt_SoakTimeStep, opt_SoakTimeStep/s_max_velocity[0]); //最大初速度が大きいほど細かく更新を行う

                for (float a = 0; a < opt_SoakTimeStep; a+=UV_var_t){

                    for(j=1+threadIdx.y; j<=stroke_length_y-1; j+=blockDim.y){
                        for(i=1+threadIdx.x; i<=stroke_length_x-1; i+=blockDim.x){
                            if(dev_M[SubImageIndex+i+j*stroke_length_max]==1 && dev_M[SubImageIndex+(i+1)+j*stroke_length_max]==1){
                                
                                // paper_DETAIL
                                A = powf((dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)])/2, 2)
                                    - powf((dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+2)+j*(stroke_length_max+1)])/2, 2)
                                    + (dev_u[SubImageIndex_uv+(i+1)+(j-1)*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+j*stroke_length_max]+dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max])/4
                                    - (dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max])/4;
                                B = dev_u[SubImageIndex_uv+(i+2)+j*(stroke_length_max+1)]
                                    + dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]
                                    + dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)]
                                    + dev_u[SubImageIndex_uv+(i+1)+(j-1)*(stroke_length_max+1)]
                                    - 4*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)];
                                p_grad = dev_p[SubImageIndex+i+j*stroke_length_max] - dev_p[SubImageIndex+(i+1)+j*stroke_length_max];

                                dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] + UV_var_t*(A - opt_mhu*B + p_grad - opt_kappa * exp(-0.1*(dev_p[SubImageIndex+i+j*stroke_length_max]+dev_p[SubImageIndex+(i+1)+j*stroke_length_max])/2)*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]);
                            }
                            else{
                                dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = 0; //ウェットエリア外を速度０に
                            }
                        }
                    }
                    __syncthreads();

                    for(j=1+threadIdx.y; j<=stroke_length_y-1; j+=blockDim.y){
                        for(i=1+threadIdx.x; i<=stroke_length_x-1; i+=blockDim.x){
                            if(dev_M[SubImageIndex+i+j*stroke_length_max]==1 && dev_M[SubImageIndex+i+(j+1)*stroke_length_max]==1){
                                
                                // paper_DETAIL
                                A = powf((dev_v[SubImageIndex_uv+i+j*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max])/2, 2)
                                    - powf((dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+2)*stroke_length_max])/2, 2)
                                    + (dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+i+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max])/4
                                    - (dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max])/4;
                                B = dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max]
                                    + dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max]
                                    + dev_v[SubImageIndex_uv+i+(j+2)*stroke_length_max]
                                    + dev_v[SubImageIndex_uv+i+j*stroke_length_max]
                                    - 4*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max];
                                p_grad = dev_p[SubImageIndex+i+j*stroke_length_max] - dev_p[SubImageIndex+i+(j+1)*stroke_length_max];

                                dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] + UV_var_t*(A - opt_mhu*B + p_grad - opt_kappa*exp(-0.1*(dev_p[SubImageIndex_uv+i+j*stroke_length_max]+dev_p[SubImageIndex_uv+i+(j+1)*stroke_length_max])/2)*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]);
                            }
                            else{
                                dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = 0;
                            }
                        }
                    }
                    __syncthreads();

                    //uにnew_uをコピー
                    for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                            dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                        }
                    }
                    //vにnew_vをコピー
                    for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                            dev_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_new_v[SubImageIndex_uv+j+i*stroke_length_max];
                        }
                    }
                    __syncthreads();
                }

                //////////RelaxDivergence(速度ベクトルの発散をある許容範囲τ未満になるまで緩和)//////////
                for(int n = 0; n < opt_N; n++){

                    //new_uにuをコピー
                    for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                            dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                        }
                    }
                    //new_vにvをコピー
                    for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                            dev_new_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_v[SubImageIndex_uv+j+i*stroke_length_max];
                        }
                    }
                    __syncthreads();

                    //初期化
                    if((threadIdx.x==0)&&(threadIdx.y==0)) s_delta_MAX[0] = 0;
                    __syncthreads();

                    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                            if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                                delta = opt_xi * (dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] - dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] + dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] - dev_v[SubImageIndex_uv+i+j*stroke_length_max]);
                                dev_p[SubImageIndex+i+j*stroke_length_max] =  dev_p[SubImageIndex+i+j*stroke_length_max] - delta;
                                if(i!=stroke_length_x && dev_M[SubImageIndex+(i+1)+j*stroke_length_max]==1) dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] - delta;
                                if(j!=stroke_length_y && dev_M[SubImageIndex+i+(j+1)*stroke_length_max]==1) dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] - delta;
                                atomicMaxFloat(&s_delta_MAX[0], fabsf(delta));
                            }
                        }
                    }
                    __syncthreads();

                    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                            if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                                if(i!=0 && dev_M[SubImageIndex+(i-1)+j*stroke_length_max]==1) dev_new_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] + delta;
                                if(j!=0 && dev_M[SubImageIndex+i+(j-1)*stroke_length_max]==1) dev_new_v[SubImageIndex_uv+i+j*stroke_length_max] = dev_new_v[SubImageIndex_uv+i+j*stroke_length_max] + delta;
                            }
                        }
                    }
                    __syncthreads();

                    if(s_delta_MAX[0] < opt_tau) break;

                    //uにnew_uをコピー
                    for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                            dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                        }
                    }
                    //vにnew_vをコピー
                    for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                        for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                            dev_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_new_v[SubImageIndex_uv+j+i*stroke_length_max];
                        }
                    }
                }

                //////////FlowOutward/////////
                for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
                    for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                        dev_gauss_M[SubImageIndex+i+j*stroke_length_max] = 0; //dev_gauss_Mを0で初期化
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        //注目しているピクセルがウェットエリアならガウスフィルタによる拡散された値を周囲に足し込む
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            sum = 0;
                            filter_sum = 1;
                            for(l=-c; l<=c; l++){
                                for(k=-c; k<=c; k++){
                                    //フィルタの端ピクセルがない場合、分子には加算せず分母から減算
                                    if( (i+k)<0 || (i+k)>stroke_length_x || (j+l)<0 || (j+l)>stroke_length_y){
                                        filter_sum -= dev_gauss_filter[(k+c)+(l+c)*w];
                                    }else{
                                        sum += dev_M[SubImageIndex+(i+k)+(j+l)*stroke_length_max] * dev_gauss_filter[(k+c)+(l+c)*w];
                                    }
                                }
                            }
                            dev_gauss_M[SubImageIndex+i+j*stroke_length_max] = sum / filter_sum;
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_p[SubImageIndex+i+j*stroke_length_max] = dev_p[SubImageIndex+i+j*stroke_length_max] - opt_eta * opt_SoakTimeStep * (1-dev_gauss_M[SubImageIndex+i+j*stroke_length_max])*dev_M[SubImageIndex+i+j*stroke_length_max];
                        }
                    }
                }
                __syncthreads();

                //////////MovePigment//////////
                for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
                    for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                        dev_new_gR[SubImageIndex+i+j*stroke_length_max] = dev_gR[SubImageIndex+i+j*stroke_length_max]; //コピー
                        dev_new_gG[SubImageIndex+i+j*stroke_length_max] = dev_gG[SubImageIndex+i+j*stroke_length_max]; //コピー
                        dev_new_gB[SubImageIndex+i+j*stroke_length_max] = dev_gB[SubImageIndex+i+j*stroke_length_max]; //コピー
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_new_gR[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gR[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gG[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gG[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gB[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gB[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_new_gR[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gR[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gG[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gG[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gB[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gB[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_new_gR[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gR[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gG[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gG[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gB[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gB[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_new_gR[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gR[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gG[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gG[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gB[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gB[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                            dev_new_gR[SubImageIndex+i+j*stroke_length_max] = dev_new_gR[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gG[SubImageIndex+i+j*stroke_length_max] = dev_new_gG[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_new_gB[SubImageIndex+i+j*stroke_length_max] = dev_new_gB[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                        }
                    }
                }
                __syncthreads();

                for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
                    for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                        dev_gR[SubImageIndex+i+j*stroke_length_max] = dev_new_gR[SubImageIndex+i+j*stroke_length_max]; //コピー
                        dev_gG[SubImageIndex+i+j*stroke_length_max] = dev_new_gG[SubImageIndex+i+j*stroke_length_max]; //コピー
                        dev_gB[SubImageIndex+i+j*stroke_length_max] = dev_new_gB[SubImageIndex+i+j*stroke_length_max]; //コピー
                    }
                }
                __syncthreads();

                //////////TransferPigment//////////
                for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                    for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                        if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){

                            if(opt_USE_DETAIL_TP){
                                down_ratio = opt_SoakTimeStep * (1-dev_h[SubImageIndex+i+j*stroke_length_max]*opt_exposure) * opt_deposit;
                                up_ratio = opt_SoakTimeStep * (1+(dev_h[SubImageIndex+i+j*stroke_length_max]-1)*opt_exposure) * opt_lift;
                            }else{
                                down_ratio = opt_SoakTimeStep * (1-dev_h[SubImageIndex+i+j*stroke_length_max]*opt_gamma) * opt_rho;
                                up_ratio = opt_SoakTimeStep * (1+(dev_h[SubImageIndex+i+j*stroke_length_max]-1)*opt_gamma) * opt_rho / opt_omega;
                            }

                            //R
                            down = dev_gR[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                            up   = dev_dR[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                            if(dev_dR[SubImageIndex+i+j*stroke_length_max]+down > 1)
                                down = fmaxf(0,1-dev_dR[SubImageIndex+i+j*stroke_length_max]);
                            if(dev_gR[SubImageIndex+i+j*stroke_length_max]+up > 1)
                                up = fmaxf(0,1-dev_gR[SubImageIndex+i+j*stroke_length_max]);
                            dev_dR[SubImageIndex+i+j*stroke_length_max] = dev_dR[SubImageIndex+i+j*stroke_length_max]+down-up;
                            dev_gR[SubImageIndex+i+j*stroke_length_max] = dev_gR[SubImageIndex+i+j*stroke_length_max]+up-down;

                            //G
                            down = dev_gG[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                            up   = dev_dG[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                            if(dev_dG[SubImageIndex+i+j*stroke_length_max]+down > 1)
                                down = fmaxf(0,1-dev_dG[SubImageIndex+i+j*stroke_length_max]);
                            if(dev_gG[SubImageIndex+i+j*stroke_length_max]+up > 1)
                                up = fmaxf(0,1-dev_gG[SubImageIndex+i+j*stroke_length_max]);
                            dev_dG[SubImageIndex+i+j*stroke_length_max] = dev_dG[SubImageIndex+i+j*stroke_length_max]+down-up;
                            dev_gG[SubImageIndex+i+j*stroke_length_max] = dev_gG[SubImageIndex+i+j*stroke_length_max]+up-down;

                            //B
                            down = dev_gB[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                            up   = dev_dB[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                            if(dev_dB[SubImageIndex+i+j*stroke_length_max]+down > 1)
                                down = fmaxf(0,1-dev_dB[SubImageIndex+i+j*stroke_length_max]);
                            if(dev_gB[SubImageIndex+i+j*stroke_length_max]+up > 1)
                                up = fmaxf(0,1-dev_gB[SubImageIndex+i+j*stroke_length_max]);
                            dev_dB[SubImageIndex+i+j*stroke_length_max] = dev_dB[SubImageIndex+i+j*stroke_length_max]+down-up;
                            dev_gB[SubImageIndex+i+j*stroke_length_max] = dev_gB[SubImageIndex+i+j*stroke_length_max]+up-down;
                        }
                    }
                }
            }
            __syncthreads();

            //if((threadIdx.x==0)&&(threadIdx.y==0))    printf("(%d,%d)Time_Roop_end\n",x,y);
            //__syncthreads();

            //堆積顔料をRGBに変換しキャンバスに描画
            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){          

                    // シミュレーション終了時に水中の顔料を落とすか拭き取るか
                    if(opt_FloatPigmentOnPaper){
                        s_test_Canvas_R[i+j*stroke_length_max] = (1 - dev_dR[SubImageIndex+i+j*stroke_length_max]) * 255;    //CMY[0,1]->RGB[0,255]
                        s_test_Canvas_G[i+j*stroke_length_max] = (1 - dev_dG[SubImageIndex+i+j*stroke_length_max]) * 255;
                        s_test_Canvas_B[i+j*stroke_length_max] = (1 - dev_dB[SubImageIndex+i+j*stroke_length_max]) * 255;
                    }
                    else{
                        //R
                        pigment_density = dev_gR[SubImageIndex+i+j*stroke_length_max] + dev_dR[SubImageIndex+i+j*stroke_length_max];
                        LIMIT_RANGE(pigment_density, 0, 1);
                        s_test_Canvas_R[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_R[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_R[x+y*width];

                        //G
                        pigment_density = dev_gG[SubImageIndex+i+j*stroke_length_max] + dev_dG[SubImageIndex+i+j*stroke_length_max];
                        LIMIT_RANGE(pigment_density, 0, 1);
                        s_test_Canvas_G[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_G[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_G[x+y*width];

                        //B
                        pigment_density = dev_gB[SubImageIndex+i+j*stroke_length_max] + dev_dB[SubImageIndex+i+j*stroke_length_max];
                        LIMIT_RANGE(pigment_density, 0, 1);
                        s_test_Canvas_B[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_B[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_B[x+y*width];
                    }
                }
            }
            __syncthreads();

            //if((threadIdx.x==0)&&(threadIdx.y==0))    printf("(%d,%d)Paint_Stroke_end\n",x,y);
            //__syncthreads();

            //実際にストロークを描いた時の誤差を計算
            if((threadIdx.x==0)&&(threadIdx.y==0)){
                s_improved_value[0] = 0; //初期化
            }
            __syncthreads();
            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                        atomicAdd(&s_improved_value[0], abs(dev_cmprR[(left_end+i)+(upper_end+j)*width]-dev_nimgR[(left_end+i)+(upper_end+j)*width]) - abs(dev_cmprR[(left_end+i)+(upper_end+j)*width]-s_test_Canvas_R[i+j*stroke_length_max]));
                        atomicAdd(&s_improved_value[0], abs(dev_cmprG[(left_end+i)+(upper_end+j)*width]-dev_nimgG[(left_end+i)+(upper_end+j)*width]) - abs(dev_cmprG[(left_end+i)+(upper_end+j)*width]-s_test_Canvas_G[i+j*stroke_length_max]));
                        atomicAdd(&s_improved_value[0], abs(dev_cmprB[(left_end+i)+(upper_end+j)*width]-dev_nimgB[(left_end+i)+(upper_end+j)*width]) - abs(dev_cmprB[(left_end+i)+(upper_end+j)*width]-s_test_Canvas_B[i+j*stroke_length_max]));
                    }
                }
            }
            __syncthreads();

            //if((threadIdx.x==0)&&(threadIdx.y==0))    printf("(%d,%d)Error_cul_end\n",x,y);
            //__syncthreads();

            if((threadIdx.x==0)&&(threadIdx.y==0)){
                dev_GLOBAL_improved_value_map[x+y*width] = s_improved_value[0];
                //printf("(%d,%d) = %d\n",x,y,s_improved_value[0]);
            }
            __syncthreads();
        }
    }
}

//改善値マップ中の最大値を探索する関数
__global__ void gpu_select_best_stroke(int *dev_GLOBAL_improved_value_map, int *dev_best_x, int *dev_best_y, int *dev_diff_stroke_max, int width, int height){

    dev_best_x[0] = 0;
    dev_best_y[0] = 0;
    dev_diff_stroke_max[0] = -99999999;

    for(int y=0; y<height; y++){
        for(int x=0; x<width; x++){
            if(dev_GLOBAL_improved_value_map[x+y*width] > dev_diff_stroke_max[0]){
                dev_best_x[0] = x;
                dev_best_y[0] = y;
                dev_diff_stroke_max[0] = dev_GLOBAL_improved_value_map[x+y*width];
            }
        }
    }
}

//実際にベストストロークを描画する関数
__global__ void gpu_draw_best_stroke(float *dev_PerlinNoise, int *dev_nimgR, int *dev_nimgG, int *dev_nimgB, int *dev_best_stroke_map_pnum, float *dev_best_stroke_map_point_x, float *dev_best_stroke_map_point_y,
                                int *dev_best_stroke_map_R, int *dev_best_stroke_map_G, int *dev_best_stroke_map_B,float *dev_grad_hx, float *dev_grad_hy, char *dev_M, float *dev_u, float *dev_new_u,
                                float *dev_v, float *dev_new_v, float *dev_p, float *dev_gR, float *dev_gG, float *dev_gB, float *dev_dR, float *dev_dG, float *dev_dB, float *dev_new_gR, float *dev_new_gG,
                                float *dev_new_gB, float *dev_new_dR, float *dev_new_dG, float *dev_new_dB, float *dev_gauss_filter, float *dev_gauss_M, float *dev_h, int *dev_best_x, int *dev_best_y, 
                                int width, int height, int t){

    int global_blockID = blockIdx.x + blockIdx.y * gridDim.x; //1ブロックしか起動しないのでglobal_blockID=0
    int stroke_length_max = opt_thick_max*(opt_max_stroke+2);
    int SubImageIndex = global_blockID * stroke_length_max * stroke_length_max;
    int SubImageIndex_uv = global_blockID * stroke_length_max * (stroke_length_max+1);

    int x,y,i,j,k,l;
    int pnum, stroke_partition, left_end, right_end, upper_end, lower_end, stroke_length_x, stroke_length_y;
    float scale, temp_x, temp_y, tmpSP_start_x, tmpSP_start_y, tmpSP_end_x, tmpSP_end_y, tmpSP0_x, tmpSP0_y, tmpSP1_x, tmpSP1_y;
    float UV_var_t, A, B, p_grad, delta, sum, filter_sum, down, up, down_ratio, up_ratio, pigment_density;
    
    int tmp_density_R = 255 - 255 * opt_ratio;//描画色の濃度
    int tmp_density_G = 255 - 255 * opt_ratio;//描画色の濃度
    int tmp_density_B = 255 - 255 * opt_ratio;//描画色の濃度

    int w = (int)(ceil(3.0*opt_K/6.0+0.5)*2-1); //とりあえず動く計算
    int c = (w-1)/2;

    __shared__ short s_test_Canvas_R[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ short s_test_Canvas_G[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ short s_test_Canvas_B[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ float s_max_velocity[1];
    __shared__ float s_delta_MAX[1];

    x = dev_best_x[0];
    y = dev_best_y[0];
    pnum = dev_best_stroke_map_pnum[x+y*width];

    //////////Paint_Water_Stroke(試しに描いてみて誤差を確認)//////////

    //ストローク点を囲む端の座標を特定
    left_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0];       //切り捨て
    right_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]+1;    //切り上げ
    upper_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0];      //切り捨て
    lower_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]+1;    //切り上げ
    for(i=1; i<pnum; i++){
        if(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i] < left_end) left_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i];
        if(right_end < dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]) right_end = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]+1;
        if(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i] < upper_end) upper_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i];
        if(lower_end < dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]) lower_end = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]+1;
    }

    //ストローク半径分、端座標を膨張
    left_end-=t; right_end+=t; upper_end-=t; lower_end+=t;
    if(left_end < 0) left_end = 0;
    if(width < right_end) right_end = width-1;
    if(upper_end < 0) upper_end=0;
    if(height < lower_end) lower_end = height-1;

    stroke_length_x = right_end - left_end; //ストロークの横の長さ
    stroke_length_y = lower_end - upper_end; //ストロークの縦の長さ

    //各パラメータの初期化
    for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
        for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
            dev_M[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_p[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_gR[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_gG[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_gB[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_gR[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_gG[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_gB[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_dR[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_dG[SubImageIndex+j+i*stroke_length_max] = 0;
            dev_new_dB[SubImageIndex+j+i*stroke_length_max] = 0;
        }
    }

    //uとvを初期化
    for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
        for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
            dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = 0;
            dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = 0;
        }
    }
    for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
        for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
            dev_v[SubImageIndex_uv+j+i*stroke_length_max] = 0;
            dev_new_v[SubImageIndex_uv+j+i*stroke_length_max] = 0;
        }
    }

    //影響箇所をコピー
    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
            dev_dR[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgR[(left_end+i)+(upper_end+j)*width] / 255.0;
            dev_dG[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgG[(left_end+i)+(upper_end+j)*width] / 255.0;
            dev_dB[SubImageIndex+i+j*stroke_length_max] = 1 - dev_nimgB[(left_end+i)+(upper_end+j)*width] / 255.0;
            s_test_Canvas_R[i+j*stroke_length_max] = dev_nimgR[(left_end+i)+(upper_end+j)*width];
            s_test_Canvas_G[i+j*stroke_length_max] = dev_nimgG[(left_end+i)+(upper_end+j)*width];
            s_test_Canvas_B[i+j*stroke_length_max] = dev_nimgB[(left_end+i)+(upper_end+j)*width];
            dev_h[SubImageIndex+i+j*stroke_length_max] = dev_PerlinNoise[(left_end+i)+(upper_end+j)*width];
        }
    }
    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
            dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] = (-1)*dev_grad_hx[(left_end+i)+(upper_end+j)*(width+1)];
        }
    }
    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
            dev_v[SubImageIndex_uv+i+j*stroke_length_max] = (-1)*dev_grad_hy[(left_end+i)+(upper_end+j)*width];
        }
    }
    __syncthreads();

    //////////set_WetStroke(ストローク点に従いウェットエリアと水量を計算)//////////
    if(pnum == 2){//もし制御点が2つの時は直線を引く
        stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1])
                            + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]); //線分の分割数                    
        
        for(k=0; k<=stroke_partition; k++){
            scale = (float)k / stroke_partition;
            temp_x = dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] + (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0])*scale;
            temp_y = dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] + (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0])*scale;

            for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                    if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                        if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                            dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                            dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                            dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                            dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                            dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                        }
                    }
                }
            }
        }
    
    //制御点が3つ以上の場合
    }else{

        //1.最初の制御点について
        //端の一つ外の制御点を適当に決める
        tmpSP_start_x = 2*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
        tmpSP_start_y = 2*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];
        
        //ベジエ曲線用の中間点を2点決める
        tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1]-tmpSP_start_x)/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0];
        tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]-tmpSP_start_y)/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0];
        tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+2])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
        tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+2])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];

        //線分の分割数
        stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1])
                        + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1]);
        
        for(k=0; k<=stroke_partition; k++){
            scale = (float)k / stroke_partition;
            temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+0] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+1];
            temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+0] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+1];

            for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                    if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                        if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                            dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                            dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                            dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                            dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                            dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                        }
                    }
                }
            }
        }
        __syncthreads();

        //2.中間の制御点について
        for(i=1; i<pnum-2; i++){

            //ベジエ曲線用の中間点を2点決める
            tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i-1])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i];
            tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i-1])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i];
            tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+2])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1];
            tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+2])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1];

            //線分の分割数
            stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1])
                            + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1]);

            for(j=0; j<=stroke_partition; j++){
                scale = (float)j / stroke_partition;
                temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+i+1];
                temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+i+1];

                for(k=threadIdx.y+temp_y-t-upper_end; k<=temp_y+t-upper_end; k+=blockDim.y){
                    for(l=threadIdx.x+temp_x-t-left_end; l<=temp_x+t-left_end; l+=blockDim.x){
                        if((l>=0) && (l<=stroke_length_x) && (k>=0) && (k<=stroke_length_y)){//画像内かどうか
                            if((l-temp_x+left_end)*(l-temp_x+left_end)+(k-temp_y+upper_end)*(k-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                                dev_M[SubImageIndex+l+k*stroke_length_max] = 1;
                                dev_gR[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                                dev_gG[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                                dev_gB[SubImageIndex+l+k*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                                dev_p[SubImageIndex+l+k*stroke_length_max] = 1.0;
                            }
                        }
                    }
                }
            }
        }

        //3.最後の制御点について
        //端の一つ外の制御点を適当に決める
        tmpSP_end_x = 2*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2];
        tmpSP_end_y = 2*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2];

        //ベジエ曲線用の中間点を2点決める
        tmpSP0_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-3])/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2];
        tmpSP0_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-3])/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2];
        tmpSP1_x = (dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2]-tmpSP_end_x)/6.0 + dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
        tmpSP1_y = (dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2]-tmpSP_end_y)/6.0 + dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];

        //線分の分割数
        stroke_partition = fabsf(dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2]-dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1])
                        + fabsf(dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2]-dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1]);

        for(k=0; k<=stroke_partition; k++){
            scale = (float)k / stroke_partition;
            temp_x = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-2] + 3*scale*(1-scale)*(1-scale)*tmpSP0_x + 3*scale*scale*(1-scale)*tmpSP1_x + scale*scale*scale*dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+pnum-1];
            temp_y = (1-scale)*(1-scale)*(1-scale)*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-2] + 3*scale*(1-scale)*(1-scale)*tmpSP0_y + 3*scale*scale*(1-scale)*tmpSP1_y + scale*scale*scale*dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+pnum-1];

            for(i=threadIdx.y+temp_y-t-upper_end; i<=temp_y+t-upper_end; i+=blockDim.y){
                for(j=threadIdx.x+temp_x-t-left_end; j<=temp_x+t-left_end; j+=blockDim.x){
                    if((j>=0) && (j<=stroke_length_x) && (i>=0) && (i<=stroke_length_y)){//画像内かどうか
                        if((j-temp_x+left_end)*(j-temp_x+left_end)+(i-temp_y+upper_end)*(i-temp_y+upper_end) <= t*t){//円の範囲内かどうか
                            dev_M[SubImageIndex+j+i*stroke_length_max] = 1;
                            dev_gR[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_R[x+y*width]/255.0;
                            dev_gG[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_G[x+y*width]/255.0;
                            dev_gB[SubImageIndex+j+i*stroke_length_max] = 1 - dev_best_stroke_map_B[x+y*width]/255.0;
                            dev_p[SubImageIndex+j+i*stroke_length_max] = 1.0;
                        }
                    }
                }
            }
        }
    }
    __syncthreads();

    //時間の経過を表すループ
    for (float time=0; time<opt_SoakTime; time+=opt_SoakTimeStep){

        //////////UpdateVelocities(一定時間経過後の速度変化を計算)//////////
        if((threadIdx.x==0)&&(threadIdx.y==0)) s_max_velocity[0] = 0;//最大初速度を初期化
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max]==1){
                    atomicMaxFloat(&s_max_velocity[0], fabsf(dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]));
                    atomicMaxFloat(&s_max_velocity[0], fabsf(dev_v[SubImageIndex_uv+i+j*stroke_length_max]));
                }
            }
        }
        __syncthreads();

        if(s_max_velocity[0] > opt_StopSoakVero) break; //水速度が暴走したときは停止
        UV_var_t = fminf(opt_SoakTimeStep, opt_SoakTimeStep/s_max_velocity[0]); //最大初速度が大きいほど細かく更新を行う

        for (float a = 0; a < opt_SoakTimeStep; a+=UV_var_t){

            for(j=1+threadIdx.y; j<=stroke_length_y-1; j+=blockDim.y){
                for(i=1+threadIdx.x; i<=stroke_length_x-1; i+=blockDim.x){
                    if(dev_M[SubImageIndex+i+j*stroke_length_max]==1 && dev_M[SubImageIndex+(i+1)+j*stroke_length_max]==1){
                        
                        // paper_DETAIL
                        A = powf((dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)])/2, 2)
                            - powf((dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+2)+j*(stroke_length_max+1)])/2, 2)
                            + (dev_u[SubImageIndex_uv+(i+1)+(j-1)*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+j*stroke_length_max]+dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max])/4
                            - (dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max])/4;
                        B = dev_u[SubImageIndex_uv+(i+2)+j*(stroke_length_max+1)]
                            + dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]
                            + dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)]
                            + dev_u[SubImageIndex_uv+(i+1)+(j-1)*(stroke_length_max+1)]
                            - 4*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)];
                        p_grad = dev_p[SubImageIndex+i+j*stroke_length_max] - dev_p[SubImageIndex+(i+1)+j*stroke_length_max];

                        dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] + UV_var_t*(A - opt_mhu*B + p_grad - opt_kappa * exp(-0.1*(dev_p[SubImageIndex+i+j*stroke_length_max]+dev_p[SubImageIndex+(i+1)+j*stroke_length_max])/2)*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]);
                    }
                    else{
                        dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = 0; //ウェットエリア外を速度０に
                    }
                }
            }
            __syncthreads();

            for(j=1+threadIdx.y; j<=stroke_length_y-1; j+=blockDim.y){
                for(i=1+threadIdx.x; i<=stroke_length_x-1; i+=blockDim.x){
                    if(dev_M[SubImageIndex+i+j*stroke_length_max]==1 && dev_M[SubImageIndex+i+(j+1)*stroke_length_max]==1){
                        
                        // paper_DETAIL
                        A = powf((dev_v[SubImageIndex_uv+i+j*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max])/2, 2)
                            - powf((dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+2)*stroke_length_max])/2, 2)
                            + (dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+i+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max])/4
                            - (dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]+dev_u[SubImageIndex_uv+(i+1)+(j+1)*(stroke_length_max+1)])*(dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]+dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max])/4;
                        B = dev_v[SubImageIndex_uv+(i+1)+(j+1)*stroke_length_max]
                            + dev_v[SubImageIndex_uv+(i-1)+(j+1)*stroke_length_max]
                            + dev_v[SubImageIndex_uv+i+(j+2)*stroke_length_max]
                            + dev_v[SubImageIndex_uv+i+j*stroke_length_max]
                            - 4*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max];
                        p_grad = dev_p[SubImageIndex+i+j*stroke_length_max] - dev_p[SubImageIndex+i+(j+1)*stroke_length_max];

                        dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] + UV_var_t*(A - opt_mhu*B + p_grad - opt_kappa*exp(-0.1*(dev_p[SubImageIndex_uv+i+j*stroke_length_max]+dev_p[SubImageIndex_uv+i+(j+1)*stroke_length_max])/2)*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]);
                    }
                    else{
                        dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = 0;
                    }
                }
            }
            __syncthreads();

            //uにnew_uをコピー
            for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                    dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                }
            }
            //vにnew_vをコピー
            for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                    dev_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_new_v[SubImageIndex_uv+j+i*stroke_length_max];
                }
            }
            __syncthreads();
        }

        //////////RelaxDivergence(速度ベクトルの発散をある許容範囲τ未満になるまで緩和)//////////
        for(int n = 0; n < opt_N; n++){

            //new_uにuをコピー
            for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                    dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                }
            }
            //new_vにvをコピー
            for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                    dev_new_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_v[SubImageIndex_uv+j+i*stroke_length_max];
                }
            }
            __syncthreads();

            //初期化
            if((threadIdx.x==0)&&(threadIdx.y==0)) s_delta_MAX[0] = 0;
            __syncthreads();

            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                        delta = opt_xi * (dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] - dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] + dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] - dev_v[SubImageIndex_uv+i+j*stroke_length_max]);
                        dev_p[SubImageIndex+i+j*stroke_length_max] =  dev_p[SubImageIndex+i+j*stroke_length_max] - delta;
                        if(i!=stroke_length_x && dev_M[SubImageIndex+(i+1)+j*stroke_length_max]==1) dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)] - delta;
                        if(j!=stroke_length_y && dev_M[SubImageIndex+i+(j+1)*stroke_length_max]==1) dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] = dev_new_v[SubImageIndex_uv+i+(j+1)*stroke_length_max] - delta;
                        atomicMaxFloat(&s_delta_MAX[0], fabsf(delta));
                    }
                }
            }
            __syncthreads();

            for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
                for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                    if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                        if(i!=0 && dev_M[SubImageIndex+(i-1)+j*stroke_length_max]==1) dev_new_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+i+j*(stroke_length_max+1)] + delta;
                        if(j!=0 && dev_M[SubImageIndex+i+(j-1)*stroke_length_max]==1) dev_new_v[SubImageIndex_uv+i+j*stroke_length_max] = dev_new_v[SubImageIndex_uv+i+j*stroke_length_max] + delta;
                    }
                }
            }
            __syncthreads();

            if(s_delta_MAX[0] < opt_tau) break;

            //uにnew_uをコピー
            for(i=threadIdx.y; i<stroke_length_max; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max+1; j+=blockDim.x){
                    dev_u[SubImageIndex_uv+j+i*(stroke_length_max+1)] = dev_new_u[SubImageIndex_uv+j+i*(stroke_length_max+1)];
                }
            }
            //vにnew_vをコピー
            for(i=threadIdx.y; i<stroke_length_max+1; i+=blockDim.y){
                for(j=threadIdx.x; j<stroke_length_max; j+=blockDim.x){
                    dev_v[SubImageIndex_uv+j+i*stroke_length_max] = dev_new_v[SubImageIndex_uv+j+i*stroke_length_max];
                }
            }
        }

        //////////FlowOutward/////////
        for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
            for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                dev_gauss_M[SubImageIndex+i+j*stroke_length_max] = 0; //dev_gauss_Mを0で初期化
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                //注目しているピクセルがウェットエリアならガウスフィルタによる拡散された値を周囲に足し込む
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    sum = 0;
                    filter_sum = 1;
                    for(l=-c; l<=c; l++){
                        for(k=-c; k<=c; k++){
                            //フィルタの端ピクセルがない場合、分子には加算せず分母から減算
                            if( (i+k)<0 || (i+k)>stroke_length_x || (j+l)<0 || (j+l)>stroke_length_y){
                                filter_sum -= dev_gauss_filter[(k+c)+(l+c)*w];
                            }else{
                                sum += dev_M[SubImageIndex+(i+k)+(j+l)*stroke_length_max] * dev_gauss_filter[(k+c)+(l+c)*w];
                            }
                        }
                    }
                    dev_gauss_M[SubImageIndex+i+j*stroke_length_max] = sum / filter_sum;
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_p[SubImageIndex+i+j*stroke_length_max] = dev_p[SubImageIndex+i+j*stroke_length_max] - opt_eta * opt_SoakTimeStep * (1-dev_gauss_M[SubImageIndex+i+j*stroke_length_max])*dev_M[SubImageIndex+i+j*stroke_length_max];
                }
            }
        }
        __syncthreads();

        //////////MovePigment//////////
        for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
            for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                dev_new_gR[SubImageIndex+i+j*stroke_length_max] = dev_gR[SubImageIndex+i+j*stroke_length_max]; //コピー
                dev_new_gG[SubImageIndex+i+j*stroke_length_max] = dev_gG[SubImageIndex+i+j*stroke_length_max]; //コピー
                dev_new_gB[SubImageIndex+i+j*stroke_length_max] = dev_gB[SubImageIndex+i+j*stroke_length_max]; //コピー
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_new_gR[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gR[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gG[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gG[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gB[SubImageIndex+(i+1)+j*stroke_length_max] = dev_new_gB[SubImageIndex+(i+1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_new_gR[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gR[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gG[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gG[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gB[SubImageIndex+(i-1)+j*stroke_length_max] = dev_new_gB[SubImageIndex+(i-1)+j*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_new_gR[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gR[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gG[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gG[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gB[SubImageIndex+i+(j+1)*stroke_length_max] = dev_new_gB[SubImageIndex+i+(j+1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_new_gR[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gR[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gG[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gG[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gB[SubImageIndex+i+(j-1)*stroke_length_max] = dev_new_gB[SubImageIndex+i+(j-1)*stroke_length_max] + fmaxf(0, opt_SoakTimeStep*(-1)*dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                    dev_new_gR[SubImageIndex+i+j*stroke_length_max] = dev_new_gR[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gG[SubImageIndex+i+j*stroke_length_max] = dev_new_gG[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_new_gB[SubImageIndex+i+j*stroke_length_max] = dev_new_gB[SubImageIndex+i+j*stroke_length_max] - fmaxf(0, opt_SoakTimeStep*dev_u[SubImageIndex_uv+(i+1)+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_u[SubImageIndex_uv+i+j*(stroke_length_max+1)]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*dev_v[SubImageIndex_uv+i+(j+1)*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]) - fmaxf(0, opt_SoakTimeStep*-dev_v[SubImageIndex_uv+i+j*stroke_length_max]*dev_gB[SubImageIndex+i+j*stroke_length_max]);
                }
            }
        }
        __syncthreads();

        for(j=threadIdx.y; j<stroke_length_max; j+=blockDim.y){
            for(i=threadIdx.x; i<stroke_length_max; i+=blockDim.x){
                dev_gR[SubImageIndex+i+j*stroke_length_max] = dev_new_gR[SubImageIndex+i+j*stroke_length_max]; //コピー
                dev_gG[SubImageIndex+i+j*stroke_length_max] = dev_new_gG[SubImageIndex+i+j*stroke_length_max]; //コピー
                dev_gB[SubImageIndex+i+j*stroke_length_max] = dev_new_gB[SubImageIndex+i+j*stroke_length_max]; //コピー
            }
        }
        __syncthreads();

        //////////TransferPigment//////////
        for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
            for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
                if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){

                    if(opt_USE_DETAIL_TP){
                        down_ratio = opt_SoakTimeStep * (1-dev_h[SubImageIndex+i+j*stroke_length_max]*opt_exposure) * opt_deposit;
                        up_ratio = opt_SoakTimeStep * (1+(dev_h[SubImageIndex+i+j*stroke_length_max]-1)*opt_exposure) * opt_lift;
                    }else{
                        down_ratio = opt_SoakTimeStep * (1-dev_h[SubImageIndex+i+j*stroke_length_max]*opt_gamma) * opt_rho;
                        up_ratio = opt_SoakTimeStep * (1+(dev_h[SubImageIndex+i+j*stroke_length_max]-1)*opt_gamma) * opt_rho / opt_omega;
                    }

                    //R
                    down = dev_gR[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                    up   = dev_dR[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                    if(dev_dR[SubImageIndex+i+j*stroke_length_max]+down > 1)
                        down = fmaxf(0,1-dev_dR[SubImageIndex+i+j*stroke_length_max]);
                    if(dev_gR[SubImageIndex+i+j*stroke_length_max]+up > 1)
                        up = fmaxf(0,1-dev_gR[SubImageIndex+i+j*stroke_length_max]);
                    dev_dR[SubImageIndex+i+j*stroke_length_max] = dev_dR[SubImageIndex+i+j*stroke_length_max]+down-up;
                    dev_gR[SubImageIndex+i+j*stroke_length_max] = dev_gR[SubImageIndex+i+j*stroke_length_max]+up-down;

                    //G
                    down = dev_gG[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                    up   = dev_dG[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                    if(dev_dG[SubImageIndex+i+j*stroke_length_max]+down > 1)
                        down = fmaxf(0,1-dev_dG[SubImageIndex+i+j*stroke_length_max]);
                    if(dev_gG[SubImageIndex+i+j*stroke_length_max]+up > 1)
                        up = fmaxf(0,1-dev_gG[SubImageIndex+i+j*stroke_length_max]);
                    dev_dG[SubImageIndex+i+j*stroke_length_max] = dev_dG[SubImageIndex+i+j*stroke_length_max]+down-up;
                    dev_gG[SubImageIndex+i+j*stroke_length_max] = dev_gG[SubImageIndex+i+j*stroke_length_max]+up-down;

                    //B
                    down = dev_gB[SubImageIndex+i+j*stroke_length_max] * down_ratio;
                    up   = dev_dB[SubImageIndex+i+j*stroke_length_max] * up_ratio;
                    if(dev_dB[SubImageIndex+i+j*stroke_length_max]+down > 1)
                        down = fmaxf(0,1-dev_dB[SubImageIndex+i+j*stroke_length_max]);
                    if(dev_gB[SubImageIndex+i+j*stroke_length_max]+up > 1)
                        up = fmaxf(0,1-dev_gB[SubImageIndex+i+j*stroke_length_max]);
                    dev_dB[SubImageIndex+i+j*stroke_length_max] = dev_dB[SubImageIndex+i+j*stroke_length_max]+down-up;
                    dev_gB[SubImageIndex+i+j*stroke_length_max] = dev_gB[SubImageIndex+i+j*stroke_length_max]+up-down;
                }
            }
        }
    }
    __syncthreads();

    //堆積顔料をRGBに変換しキャンバスに描画
    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){          

            // シミュレーション終了時に水中の顔料を落とすか拭き取るか
            if(opt_FloatPigmentOnPaper){
                s_test_Canvas_R[i+j*stroke_length_max] = (1 - dev_dR[SubImageIndex+i+j*stroke_length_max]) * 255;    //CMY[0,1]->RGB[0,255]
                s_test_Canvas_G[i+j*stroke_length_max] = (1 - dev_dG[SubImageIndex+i+j*stroke_length_max]) * 255;
                s_test_Canvas_B[i+j*stroke_length_max] = (1 - dev_dB[SubImageIndex+i+j*stroke_length_max]) * 255;
            }
            else{
                //R
                pigment_density = dev_gR[SubImageIndex+i+j*stroke_length_max] + dev_dR[SubImageIndex+i+j*stroke_length_max];
                LIMIT_RANGE(pigment_density, 0, 1);
                s_test_Canvas_R[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_R[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_R[x+y*width];

                //G
                pigment_density = dev_gG[SubImageIndex+i+j*stroke_length_max] + dev_dG[SubImageIndex+i+j*stroke_length_max];
                LIMIT_RANGE(pigment_density, 0, 1);
                s_test_Canvas_G[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_G[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_G[x+y*width];

                //B
                pigment_density = dev_gB[SubImageIndex+i+j*stroke_length_max] + dev_dB[SubImageIndex+i+j*stroke_length_max];
                LIMIT_RANGE(pigment_density, 0, 1);
                s_test_Canvas_B[i+j*stroke_length_max] = (1-pigment_density) * s_test_Canvas_B[i+j*stroke_length_max] + pigment_density * dev_best_stroke_map_B[x+y*width];
            }
        }
    }
    __syncthreads();

    for(j=threadIdx.y; j<=stroke_length_y; j+=blockDim.y){
        for(i=threadIdx.x; i<=stroke_length_x; i+=blockDim.x){
            if(dev_M[SubImageIndex+i+j*stroke_length_max] == 1){
                //キャンバスに部分画像をコピー
                dev_nimgR[(left_end+i)+(upper_end+j)*width] = s_test_Canvas_R[i+j*stroke_length_max];
                dev_nimgG[(left_end+i)+(upper_end+j)*width] = s_test_Canvas_G[i+j*stroke_length_max];
                dev_nimgB[(left_end+i)+(upper_end+j)*width] = s_test_Canvas_B[i+j*stroke_length_max];
            }
        }
    }
    __syncthreads();
}

__global__ void gpu_reset_improved_value_map(int *dev_GLOBAL_improved_value_map, int *dev_best_stroke_map_pnum, float *dev_best_stroke_map_point_x, float *dev_best_stroke_map_point_y, int *dev_best_x, int *dev_best_y, int width, int height, int t){

    int distance;
    int best_x = dev_best_x[0];
    int best_y = dev_best_y[0];
    int pnum = dev_best_stroke_map_pnum[best_x+best_y*width];

    //ストローク点を囲む端の座標を特定
    int left_end = dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+0];
    int right_end = dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+0];
    int upper_end = dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+0];
    int lower_end = dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+0];
    for(int i=1; i<pnum; i++){
        if(dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i] < left_end) left_end = dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i];
        if(right_end < dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i]) right_end = dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i];
        if(dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i] < upper_end) upper_end = dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i];
        if(lower_end < dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i]) lower_end = dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i];
    }

    //端座標を膨張
    left_end  -= t*opt_max_stroke + t+2;
	right_end += t*opt_max_stroke + t+2;
	upper_end -= t*opt_max_stroke + t+2;
	lower_end += t*opt_max_stroke + t+2;
    if(left_end<0) left_end = 0;
	if(width <= right_end) right_end = width-1;
	if(upper_end<0) upper_end = 0;
	if(height <= lower_end) lower_end = height-1;

    for(int y=upper_end; y<=lower_end; y++) {
		for(int x=left_end; x<=right_end; x++) {
			if(dev_GLOBAL_improved_value_map[x+y*width] == UNCALCULATED) continue;
			// 描画したストローク制御点と重なるストロークのみを再計算
			for(int i=0; i<pnum; i++){
				for(int j=0; j<dev_best_stroke_map_pnum[x+y*width]; j++) {
					distance = (dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+j])
                                * (dev_best_stroke_map_point_x[(best_x+best_y*width)*opt_max_stroke+i] - dev_best_stroke_map_point_x[(x+y*width)*opt_max_stroke+j])
                                + (dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+j])
                                * (dev_best_stroke_map_point_y[(best_x+best_y*width)*opt_max_stroke+i] - dev_best_stroke_map_point_y[(x+y*width)*opt_max_stroke+j]);	// 点と点のユークリッド距離
					if(distance < 2*t*2*t){
						dev_GLOBAL_improved_value_map[x+y*width] = UNCALCULATED;
						goto RIM_loopend;
					}
				}
			}
			RIM_loopend: ;
		}
	}
}