#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/stat.h>
#include <dirent.h>
#include <omp.h>
#include "sbr.h"
#include "sbr_opt.h"
#include "water.h"

#define p(s,a) printf("%s:%d\n",s,a)

PPM *gpu_c_Illust_brush_Water_best(PPM *in, char *filename);

PGM* GLOBAL_improved_value_map;

//GPUのエラーを検出する関数
void printCudaLastError(){
	hipError_t err = hipGetLastError();
	printf("hipGetLastError::%s(code:%d)\n",hipGetErrorString(err),err);
	if(err)	exit(0);
}

int main(int argc, char *argv[])
{

	//実行時間計測用
	float total_time = 0;
	hipEvent_t total_time_start, total_time_end;
	hipEventCreate(&total_time_start);
	hipEventCreate(&total_time_end);

	my_clock();

	if(argc<2){
		fprintf(stderr, "Usage: program <inputfile> <outputfile>\n");
		exit(1);
	}

	#ifdef _OPENMP
        omp_set_num_threads(atoi(argv[3]));
    #endif

	image_t *in_img;
	PPM *in_ppm, *trans_ppm;


	char *name;
	char *ext;
	name = argv[1];

	//入力画像を読み込む
	/*
	ext = get_extension(name);
	if (strcmp("ppm", ext) == 0 || strcmp("pnm", ext) == 0) {
		in_ppm = read_ppm(name);
	} else if (strcmp("jpg", ext) == 0 || strcmp("jpeg", ext) == 0) {
		in_img = read_jpeg_file(name);
		dump_image_info(in_img);	//画像情報出力
		in_ppm = image_to_PPM(in_img);		//扱いやすいデータ構造に変換
		free_image(in_img);
	} else if (strcmp("png", ext) == 0) {
		in_img = read_png_file(name);
		dump_image_info(in_img);	//画像情報出力
		in_ppm = image_to_PPM(in_img);		//扱いやすいデータ構造に変換
		free_image(in_img);
	} else {
		printf("Plese use JPEG,PNG or PPM!\n");
		exit(1);
	}
	*/
	in_ppm = read_ppm(name);

	hipEventRecord(total_time_start, 0);//計測スタート

	//入力画像の絵画化
	trans_ppm = gpu_c_Illust_brush_Water_best(in_ppm, argv[2]);

	hipEventRecord(total_time_end, 0);//計測ストップ

	//実行時間表示
	hipEventElapsedTime(&total_time, total_time_start, total_time_end);
	printf("Total_Time:%f[ms]\n", total_time);

	//出力ファイル名に従って画像を出力
	/*
	ext = get_extension(argv[2]);
	if (strcmp("ppm", ext) == 0 || strcmp("pnm", ext) == 0) {
		if(write_ppm(argv[2], trans_ppm)){ printf("WRITE_PPM_ERROR (main)\n");}
	} else if (strcmp("jpg", ext) == 0 || strcmp("jpeg", ext) == 0) {
		if(write_jpeg_file(argv[2], PPM_to_image(trans_ppm))){ printf("WRITE JPG ERROR.");}
	} else if (strcmp("png", ext) == 0) {
		if(write_png_file(argv[2], PPM_to_image(trans_ppm))){ printf("WRITE PNG ERROR.");}
	}
	*/
	write_ppm(argv[2], trans_ppm);

	FreePPM(in_ppm);
	FreePPM(trans_ppm);

	pd("TOTAL_TIME[s]",my_clock());
	return 0;
}

//与えられたイメージからブラッシングによる絵画を作る
PPM *gpu_c_Illust_brush_Water_best(PPM *in, char *filename)
{

	//実行時間計測用
	float ftimer = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	my_clock();
	int i,j,x,y,xc,yc,t=1000,break_flag,pnum, offscrn_count;
	int window_diff_border = opt_window_diff_border; 	//ストローク位置探索のしきい値
	int color_diff_border = opt_color_diff_border;  	//描画色の差異のしきい値
	int max_stroke = opt_max_stroke;
	int min_stroke = opt_min_stroke;
	Point p[max_stroke];
	int stroke_histogram[max_stroke+1];
	for(i=0; i<max_stroke+1; i++){stroke_histogram[i]=0;}
	double ratio=opt_ratio;		//ストロークの濃度
	double theta, former_theta;
	double sigma, diff_sum, sum;
	int histogram_partition=opt_histogram_partition;
	int paint_count=0, nc=0, tc=-1;
	int loop_cont=opt_loop_cont, x_defo=0, y_defo=0;
	// int lc=0;
	// double maxValue, minValue;
	RGB bright;


	//最大小ストローク半径（自動化：画面の1/10,最大の1/10）
	int thick_max = opt_thick_max;//(in->height < in->width ? in->height : in->width)/10;
	int thick_min = opt_thick_min;//(thick_max/15 > 3 ? thick_max/15 : 3);


	//出力ファイル名のサイズを取得
	int namesize = strlen(filename)*2 + 16;
	char out_filename[namesize], log_filename[namesize], dir_path[namesize];
	char in_filename[namesize-16];
	char count_name[16];
	char log_sentence[2028] = "";
	image_t *out_png;
	//パスから入力ファイル名（拡張子含まない）を取得
	char tmp[namesize];
	char* tp;
	strcpy(in_filename, filename);
	strcpy(tmp, filename);
	strtok(tmp, "/\\");
	while((tp = strtok(NULL, "/\\")) != NULL ) {
		ps("in_filename",in_filename);
		ps("tmp",tmp);
		ps("tp",tp);
		strcpy(in_filename, tp);
	}
	strtok(in_filename, ".");
	ps("in_filename",in_filename);

	//出力するフォルダを生成しフォルダへのパスを格納、ログファイルを作成
	strcpy(tmp, filename);
	strcpy(dir_path, strtok(tmp, "."));
	if(mkdir(tmp, 0775)){ printf("FAIL TO CREATE DIRECTRY\n"); }
	strcat(dir_path, "/");
	strcpy(log_filename, dir_path);
	strcat(log_filename, in_filename);
	strcat(log_filename, ".log");

	//logデータ格納
	strcat(log_sentence, "<");
	strcat(log_sentence, in_filename);
	strcat(log_sentence, ">\r\n");
	strcat(log_sentence, "Stroke_Method:BestStroke\r\n");
	Add_dictionary_to_sentence(log_sentence, "width", in->width);
	Add_dictionary_to_sentence(log_sentence, "height", in->height);
	Add_dictionary_to_sentence(log_sentence, "thick_max", thick_max);
	Add_dictionary_to_sentence(log_sentence, "thick_min", thick_min);
	Add_dictionary_to_sentence(log_sentence, "max_stroke", max_stroke);
	Add_dictionary_to_sentence(log_sentence, "min_stroke", min_stroke);
	Add_dictionary_to_sentence(log_sentence, "window_diff_border", window_diff_border);
	Add_dictionary_to_sentence(log_sentence, "color_diff_border", color_diff_border);
	Add_dictionary_to_sentence_d(log_sentence, "ratio", ratio);
	Add_dictionary_to_sentence(log_sentence, "histogram_partition", histogram_partition);
	Add_dictionary_to_sentence(log_sentence, "loop_cont", loop_cont);
	Add_dictionary_to_sentence(log_sentence, "USE_Lab_ColorDiff", opt_USE_Lab_ColorDiff);
	Add_dictionary_to_sentence(log_sentence, "USE_calcu_color_bi", opt_USE_calcu_color_bi);
	Add_dictionary_to_sentence(log_sentence, "USE_gause_histogram", opt_USE_gause_histogram);
	Add_dictionary_to_sentence(log_sentence, "optimal_improved_value_border", opt_optimal_improved_value_border);
	Add_dictionary_to_sentence_d(log_sentence, "StrokeWindowStep", opt_StrokeWindowStep/t);
	if(opt_USE_calcu_Kmean_ColorSet){
		Add_dictionary_to_sentence(log_sentence, "Kmean_ClusterNum", opt_Kmean_ClusterNum);
	}else if(opt_USE_calcu_JIS_ColorSet){
		Add_dictionary_to_sentence(log_sentence, "JIS_ClusterNum", opt_JIS_ClusterNum);
	}
	if(opt2_thick_max){
		Add_dictionary_to_sentence(log_sentence, "thick_max[2]", opt2_thick_max);
		Add_dictionary_to_sentence(log_sentence, "thick_min[2]", opt2_thick_min);
		Add_dictionary_to_sentence(log_sentence, "min_stroke[2]", opt2_min_stroke);
		Add_dictionary_to_sentence_d(log_sentence, "ratio[2]", opt2_ratio);
		Add_dictionary_to_sentence(log_sentence, "loop_cont[2]", opt2_loop_cont);
	}
	strcat(log_sentence, "\r\n[Water Option]\r\n");
	Add_dictionary_to_sentence_d(log_sentence, "mhu", opt_mhu);
	Add_dictionary_to_sentence_d(log_sentence, "kappa", opt_kappa);
	Add_dictionary_to_sentence(log_sentence, "N", opt_N);
	Add_dictionary_to_sentence_d(log_sentence, "tau", opt_tau);
	Add_dictionary_to_sentence_d(log_sentence, "xi", opt_xi);
	Add_dictionary_to_sentence(log_sentence, "K", opt_K);
	Add_dictionary_to_sentence_d(log_sentence, "eta", opt_eta);
	if(opt_USE_DETAIL_TP){
		Add_dictionary_to_sentence_d(log_sentence, "deposit", opt_deposit);
		Add_dictionary_to_sentence_d(log_sentence, "lift", opt_lift);
		Add_dictionary_to_sentence_d(log_sentence, "exposure", opt_exposure);
	}else{
		Add_dictionary_to_sentence_d(log_sentence, "gamma", opt_gamma);
		Add_dictionary_to_sentence_d(log_sentence, "rho", opt_rho);
		Add_dictionary_to_sentence_d(log_sentence, "omega", opt_omega);
	}
	Add_dictionary_to_sentence(log_sentence, "SoakTme", opt_SoakTime);
	Add_dictionary_to_sentence_d(log_sentence, "SoakTimeStep", opt_SoakTimeStep);
	Add_dictionary_to_sentence_d(log_sentence, "perlin_freq", opt_perlin_freq);
	Add_dictionary_to_sentence(log_sentence, "perlin_depth", opt_perlin_depth);
	Add_dictionary_to_sentence_d(log_sentence, "variance_ratio", opt_variance_ratio);
	if(opt_USE_Backrun){
		Add_dictionary_to_sentence_d(log_sentence, "alpha", opt_alpha);
		Add_dictionary_to_sentence_d(log_sentence, "epsilon", opt_epsilon);
		Add_dictionary_to_sentence_d(log_sentence, "delta", opt_delta);
		Add_dictionary_to_sentence_d(log_sentence, "sigma", opt_sigma);
	}
	Add_dictionary_to_sentence(log_sentence, "RemovePigmentInWater", opt_RemovePigmentInWater);
	Add_dictionary_to_sentence(log_sentence, "FloatPigmentOnPaper", opt_FloatPigmentOnPaper);
	#ifdef _OPENMP
		Add_dictionary_to_sentence(log_sentence, "OMP_NUM_THREADS", omp_get_max_threads());
		printf("OMP_NUM_T:%d \n",(int)omp_get_max_threads());
	#endif
	Add_dictionary_to_sentence(log_sentence, "opt_GPU_Block_Num", opt_GPU_Block_Num_x*opt_GPU_Block_Num_y);
	Add_dictionary_to_sentence(log_sentence, "opt_GPU_Thread_Num", opt_GPU_Thread_Num_x*opt_GPU_Thread_Num_y);

	pn;

	//カラー画像分割
	PGM *gray = color_gray_conversion(in);
	PGM *nimgR = create_pgm(gray->width, gray->height, gray->bright);
	PGM *nimgG = create_pgm(gray->width, gray->height, gray->bright);
	PGM *nimgB = create_pgm(gray->width, gray->height, gray->bright);
	PGM *inR = create_pgm(gray->width, gray->height, gray->bright);
	PGM *inG = create_pgm(gray->width, gray->height, gray->bright);
	PGM *inB = create_pgm(gray->width, gray->height, gray->bright);
	devide_ppm(in, inR, inG, inB);
	//比較用のイメージ生成
	PGM *cmpr  = gray;
	PGM *cmprR = inR;
	PGM *cmprG = inG;
	PGM *cmprB = inB;
	//キャンバスイメージ生成
	PGM *nimgV = create_pgm(gray->width, gray->height, gray->bright); //明度のみのキャンバス（比較用）
	PPM *nimgC;
	if(opt_USE_input_progress_image){
		//描画中キャンバス画像を読み込む
		image_t *in_img;
		char name[128] = opt_progress_image_address;
		char* ext = get_extension(name);

		//if (strcmp("ppm", ext) == 0 || strcmp("pnm", ext) == 0) {
			nimgC = read_ppm(name);
		//}
		/* else if (strcmp("jpg", ext) == 0 || strcmp("jpeg", ext) == 0) {
			in_img = read_jpeg_file(name);
			dump_image_info(in_img);	//画像情報出力
			nimgC = image_to_PPM(in_img);		//扱いやすいデータ構造に変換
			free_image(in_img);
		} else if (strcmp("png", ext) == 0) {
			in_img = read_png_file(name);
			dump_image_info(in_img);	//画像情報出力
			nimgC = image_to_PPM(in_img);		//扱いやすいデータ構造に変換
			free_image(in_img);
		} else {
			printf("Plese use JPEG,PNG or PPM!\n");
			exit(1);
		}
		*/

		devide_ppm(nimgC, nimgR, nimgG, nimgB);
	} else{
		nimgC = create_ppm(in->width, in->height, in->bright); //実際に描画するキャンバス
	}
	nimgC->dataR = nimgR->data;
	nimgC->dataG = nimgG->data;
	nimgC->dataB = nimgB->data;

	PPM *test_Canvas;
	// #ifndef _OPENMP
		test_Canvas = create_ppm(in->width, in->height, in->bright);
	// #endif
	// PGM *improved_value_map = create_pgm(in->width, in->height, in->bright); //改善値マップ
	GLOBAL_improved_value_map = create_pgm(in->width, in->height, in->bright); //改善値マップ


	// sobelフィルタを適応した計算結果を予め格納しておく
	double **sobel_abs = create_dally(in->width, in->height);
	double **sobel_angle = create_dally(in->width, in->height);
	sobel_calcu(gray, sobel_abs, sobel_angle);

	//Greedyアプローチ
	int s_count,stroke_num;
	// int best_pnum;
	int diff_stroke_max[1];
	int tmp_num;
	int diff_stroke_max_ave[50]={};
	int best_x=0, best_y=0;
	// int miss_stroke_count=0;
	Stroke*** best_stroke_map = create_Stroke_ally(in->width, in->height, max_stroke);
	Point best_P;
	int optimal_improved_value_border = opt_optimal_improved_value_border;
	// Point before_P={0,0};

    double** h = perlin_img(in->width, in->height, opt_perlin_freq, opt_perlin_depth);
    double** grad_hx = create_dally(in->width+1, in->height);
    double** grad_hy = create_dally(in->width, in->height+1);
    calcu_grad_h(h, grad_hx, grad_hy, in->width, in->height);

	// KmeanカラーセットまたはJISカラーセット
	int CentLabel = 0, JISLabel = 0;
	int *num_cluster, *x_centlabel;
	int** x_centlabel_2D;
	RGB* ColorSet;
	if(opt_USE_calcu_Kmean_ColorSet){
		x_centlabel = (int*)malloc(sizeof(int) * in->width*in->height);
		num_cluster = (int*)malloc(sizeof(int) * opt_Kmean_ClusterNum);
		ColorSet = Kmeans_ImageLab3D(in, opt_Kmean_ClusterNum, 50, x_centlabel, num_cluster);
		x_centlabel_2D = ReshapeInt_1to2(x_centlabel, in->width, in->height);	//クラスタ番号は１から数えている
		free(x_centlabel);
		PPM* Kmean_Img = Visualize_KmeanImg(in, ColorSet, x_centlabel_2D);
		PPM* ColorSet_Img = Visualize_ColorSet(ColorSet, opt_Kmean_ClusterNum, num_cluster);
		strcpy(out_filename, dir_path);
		strcat(out_filename, in_filename);
		strcat(out_filename, "__Kmean");
		strcat(out_filename, ".png");
		if(write_png_file(out_filename, PPM_to_image(Kmean_Img))){ printf("WRITE PNG ERROR.");}
		strcpy(out_filename, dir_path);
		strcat(out_filename, in_filename);
		strcat(out_filename, "__ColorSet");
		strcat(out_filename, ".png");
		if(write_png_file(out_filename, PPM_to_image(ColorSet_Img))){ printf("WRITE PNG ERROR.");}
	}
	else if(opt_USE_calcu_JIS_ColorSet){
		ColorSet = create_JIS_ColorSet(opt_JIS_ClusterNum);
	}

	// Lab誤差
	Lab** in_Lab;
	float *in_Lab_L;
	float *in_Lab_a;
	float *in_Lab_b;
	if(opt_USE_Lab_ColorDiff){
		//　RGB入力画像をLabに変換した配列を用意
		RGB CanRGB;
		in_Lab = (Lab**)malloc(sizeof(Lab*)*(in->width));
		for(i=0; i<in->width; i++){
			in_Lab[i] = (Lab*)malloc(sizeof(Lab)*(in->height));
		}

		for(i=0; i<in->width; i++){
			for(j=0; j<in->height; j++){
				CanRGB.R = in->dataR[i][j];
				CanRGB.G = in->dataG[i][j];
				CanRGB.B = in->dataB[i][j];
				in_Lab[i][j] = RGB2Lab(CanRGB);
			}
		}

		in_Lab_L = (float*)malloc(sizeof(float)*in->width*in->height);
		in_Lab_a = (float*)malloc(sizeof(float)*in->width*in->height);
		in_Lab_b = (float*)malloc(sizeof(float)*in->width*in->height);

		for(i=0; i<in->width; i++){
			for(j=0; j<in->height; j++){
				in_Lab_L[i+j*in->width] = in_Lab[i][j].L;
				in_Lab_a[i+j*in->width] = in_Lab[i][j].a;
				in_Lab_b[i+j*in->width] = in_Lab[i][j].b;
			}
		}
	}

	//ガウスフィルタを生成
	double** gauce_filter;
    int w = (int)(ceil(3.0*opt_K/6.0+0.5)*2-1); //とりあえず動く計算
    int c=(w-1)/2;
    gauce_filter = create_dally(w, w);
	for(i=0;i<w;i++){
		for(j=0;j<w;j++){
			gauce_filter[i][j] = gause_func(i-c, j-c, opt_K/6.0);
			//printf("%.2f ",gauce_filter[i][j]);
		}
		//printf("\n");
	}

	//GPUの領域確保
	int stroke_length_max = opt_thick_max*(opt_max_stroke+2); //ストローク長さの最大値（本来はopt_thick_max*(opt_max_stroke+1)だが余裕を持たせている）
	int *dev_GLOBAL_improved_value_map;
	float *dev_PerlinNoise;
	float *dev_in_Lab_L;
	float *dev_in_Lab_a;
	float *dev_in_Lab_b;
	int *dev_cmpr;
	int *dev_cmprR;
	int *dev_cmprG;
	int *dev_cmprB;
	int *dev_nimgR;
	int *dev_nimgG;
	int *dev_nimgB;
	int *dev_best_stroke_map_pnum;
	float *dev_best_stroke_map_point_x;
	float *dev_best_stroke_map_point_y;
	int *dev_best_stroke_map_R;
	int *dev_best_stroke_map_G;
	int *dev_best_stroke_map_B;
	float *dev_gauss_filter;
	float *dev_sobel_abs;
	float *dev_sobel_angle;
	float *dev_grad_hx;
    float *dev_grad_hy;
	char *dev_M;
    float *dev_u;
    float *dev_new_u;
	float *dev_v;
	float *dev_new_v;
	float *dev_p;
	float *dev_gR;
	float *dev_gG;
	float *dev_gB;
	float *dev_dR;
	float *dev_dG;
	float *dev_dB;
	float *dev_new_gR;
	float *dev_new_gG;
	float *dev_new_gB;
	float *dev_new_dR;
	float *dev_new_dG;
	float *dev_new_dB;
	float *dev_gauss_M;
	float *dev_h;
	int *dev_best_x;
	int *dev_best_y;
	int *dev_diff_stroke_max;

	hipMalloc(&dev_GLOBAL_improved_value_map , in->width*in->height*sizeof(int));
	hipMalloc(&dev_PerlinNoise , in->width*in->height*sizeof(float));
	hipMalloc(&dev_in_Lab_L , in->width*in->height*sizeof(float));
	hipMalloc(&dev_in_Lab_a , in->width*in->height*sizeof(float));
	hipMalloc(&dev_in_Lab_b , in->width*in->height*sizeof(float));
	hipMalloc(&dev_cmpr , in->width*in->height*sizeof(int));
	hipMalloc(&dev_cmprR , in->width*in->height*sizeof(int));
	hipMalloc(&dev_cmprG , in->width*in->height*sizeof(int));
	hipMalloc(&dev_cmprB , in->width*in->height*sizeof(int));
	hipMalloc(&dev_nimgR , in->width*in->height*sizeof(int));
	hipMalloc(&dev_nimgG , in->width*in->height*sizeof(int));
	hipMalloc(&dev_nimgB , in->width*in->height*sizeof(int));
	hipMalloc(&dev_best_stroke_map_pnum , in->width*in->height*sizeof(int));
	hipMalloc(&dev_best_stroke_map_point_x , in->width*in->height*max_stroke*sizeof(float));
	hipMalloc(&dev_best_stroke_map_point_y , in->width*in->height*max_stroke*sizeof(float));
	hipMalloc(&dev_best_stroke_map_R , in->width*in->height*sizeof(int));
	hipMalloc(&dev_best_stroke_map_G , in->width*in->height*sizeof(int));
	hipMalloc(&dev_best_stroke_map_B , in->width*in->height*sizeof(int));
	hipMalloc(&dev_sobel_abs , in->width*in->height*sizeof(float));
	hipMalloc(&dev_sobel_angle , in->width*in->height*sizeof(float));
	hipMalloc(&dev_grad_hx , (in->width+1)*in->height*sizeof(float));
	hipMalloc(&dev_grad_hy , in->width*(in->height+1)*sizeof(float));
	hipMalloc(&dev_M , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(char));
	hipMalloc(&dev_u , (stroke_length_max+1)*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_u , (stroke_length_max+1)*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_v , stroke_length_max*(stroke_length_max+1)*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_v , stroke_length_max*(stroke_length_max+1)*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_p , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_gR , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_gG , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_gB , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_dR , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_dG , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_dB , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_gR , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_gG , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_gB , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_dR , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_dG , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_new_dB , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_gauss_filter , w*w*sizeof(float));
	hipMalloc(&dev_gauss_M , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_h , stroke_length_max*stroke_length_max*opt_GPU_Block_Num_x*opt_GPU_Block_Num_y*sizeof(float));
	hipMalloc(&dev_best_x , 1*sizeof(int));
	hipMalloc(&dev_best_y , 1*sizeof(int));
	hipMalloc(&dev_diff_stroke_max , 1*sizeof(int));

	float *PerlinNoise_1dim = ReshapeDouble_2to1(h, in->height, in->width);
	int *cmpr_1dim = ReshapeInt_2to1(cmpr->data, in->height, in->width);
	int *cmprR_1dim = ReshapeInt_2to1(cmprR->data, in->height, in->width);
	int *cmprG_1dim = ReshapeInt_2to1(cmprG->data, in->height, in->width);
	int *cmprB_1dim = ReshapeInt_2to1(cmprB->data, in->height, in->width);
	float *sobel_abs_1dim = ReshapeDouble_2to1(sobel_abs, in->height, in->width);
	float *sobel_angle_1dim = ReshapeDouble_2to1(sobel_angle, in->height, in->width);
	float *grad_hx_1dim = ReshapeDouble_2to1(grad_hx, in->height, in->width+1);
	float *grad_hy_1dim = ReshapeDouble_2to1(grad_hy, in->height+1, in->width);
	float *gauss_filter_1dim = ReshapeDouble_2to1(gauce_filter, w, w);
	
	//キャンバスを0で初期化
	int *nimgR_1dim = (int*)malloc(sizeof(int)*in->height*in->width);
	int *nimgG_1dim = (int*)malloc(sizeof(int)*in->height*in->width);
	int *nimgB_1dim = (int*)malloc(sizeof(int)*in->height*in->width);
	for(int i=0; i<in->width*in->height; i++){
		nimgR_1dim[i] = 255;
		nimgG_1dim[i] = 255;
		nimgB_1dim[i] = 255;
	}

	hipMemcpy(dev_PerlinNoise, PerlinNoise_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_Lab_L, in_Lab_L, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_Lab_a, in_Lab_a, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_in_Lab_b, in_Lab_b, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_cmpr, cmpr_1dim, in->width*in->height*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_cmprR, cmprR_1dim, in->width*in->height*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_cmprG, cmprG_1dim, in->width*in->height*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_cmprB, cmprB_1dim, in->width*in->height*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_sobel_abs, sobel_abs_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_sobel_angle, sobel_angle_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_grad_hx, grad_hx_1dim, (in->width+1)*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_grad_hy, grad_hy_1dim, in->width*(in->height+1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_nimgR, nimgR_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_nimgG, nimgG_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_nimgB, nimgB_1dim, in->width*in->height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_gauss_filter, gauss_filter_1dim, w*w*sizeof(float), hipMemcpyHostToDevice);

	dim3 block_num(opt_GPU_Block_Num_x, opt_GPU_Block_Num_y);	//カーネルの起動ブロック数を定義
	dim3 thread_num(opt_GPU_Thread_Num_x, opt_GPU_Thread_Num_y);	//カーネルの起動スレッド数を定義

	///////////////////preprocess終了/////////////////
	Add_dictionary_to_sentence_d(log_sentence, "PreProsessTIME[s]", my_clock());
	pd("PreProsessTIME[s]",my_clock());

	//太いストロークから順番にストロークを小さくしておおまかに絵の形を取っていく
	for(t=thick_max; t>=thick_min; t--){
		if(opt_num_thick){
			int thick_arr[opt_num_thick] = opt_thick_assignment;
			int thick_flag=1;
			for (i = 0; i < opt_num_thick; i++){
				if(t==thick_arr[i]) thick_flag=0;
			}
			if(thick_flag) continue;
		}

		printf("t = %d\n",t);

		// 最適なストロークに関するデータを初期化
		format_ally(GLOBAL_improved_value_map->data, GLOBAL_improved_value_map->width, GLOBAL_improved_value_map->height, UNCALCULATED);
		int *GLOBAL_improved_value_map_1dim = ReshapeInt_2to1(GLOBAL_improved_value_map->data, in->height, in->width);
		hipMemcpy(dev_GLOBAL_improved_value_map, GLOBAL_improved_value_map_1dim, in->width*in->height*sizeof(int), hipMemcpyHostToDevice);

		stroke_num = 99999;

		for(s_count=0; s_count<stroke_num; s_count++) {  //ある半径におけるストローク回数

			//hipEventRecord(start, 0);//計測スタート

			//各ストロークの改善値を並列に計算するカーネル
			gpu_calculate_best_stroke<<<block_num,thread_num>>>(dev_GLOBAL_improved_value_map, dev_PerlinNoise, dev_cmprR, dev_cmprG, dev_cmprB, dev_nimgR,
														dev_nimgG, dev_nimgB, dev_best_stroke_map_pnum, dev_best_stroke_map_point_x,
														dev_best_stroke_map_point_y, dev_best_stroke_map_R, dev_best_stroke_map_G, 
														dev_best_stroke_map_B, dev_sobel_abs, dev_sobel_angle, dev_grad_hx, dev_grad_hy, dev_in_Lab_L,
														dev_in_Lab_a, dev_in_Lab_b, dev_M, dev_u, dev_new_u,
														dev_v, dev_new_v, dev_p, dev_gR, dev_gG, dev_gB, dev_dR, dev_dG, dev_dB,
														dev_new_gR, dev_new_gG, dev_new_gB, dev_new_dR, dev_new_dG, dev_new_dB,
														dev_gauss_filter, dev_gauss_M, dev_h, in->width, in->height, t);
			hipDeviceSynchronize();
			//printCudaLastError();

			//改善値マップ中の最大値を探索するカーネル
			gpu_select_best_stroke<<<1,1>>>(dev_GLOBAL_improved_value_map, dev_best_x, dev_best_y, dev_diff_stroke_max, in->width, in->height);
			hipDeviceSynchronize();
			//printCudaLastError();

			//改善値マップ中の最大値をCPUにコピー
			hipMemcpy(diff_stroke_max, dev_diff_stroke_max, 1*sizeof(int), hipMemcpyDeviceToHost);
			printf("%d : diff_stroke_max = %d\n" , s_count, diff_stroke_max[0]);

			//改善値マップ中の最大値が閾値以下ならば次の半径の処理へ進む
			if(diff_stroke_max[0] < optimal_improved_value_border) {
				strcat(log_sentence, "\r\n");
				Add_dictionary_to_sentence(log_sentence, "t", t);
				Add_dictionary_to_sentence(log_sentence, "s_count", s_count);
				Add_dictionary_to_sentence_d(log_sentence, "TIME[s]", my_clock());
				{
					strcpy(out_filename, dir_path);
					strcat(out_filename, in_filename);
					snprintf(count_name, 16, "%02d", t);
					strcat(out_filename, "_t");
					strcat(out_filename, count_name);
					strcat(out_filename, "_BSM.pgm");
					if(write_pgm(out_filename, GLOBAL_improved_value_map)){ printf("WRITE PGM ERROR.");}
					printf("%s\n",out_filename);
				}
				break;
			}

			//実際にストロークを描画する関数
			gpu_draw_best_stroke<<<1,thread_num>>>(dev_PerlinNoise, dev_nimgR, dev_nimgG, dev_nimgB, dev_best_stroke_map_pnum, dev_best_stroke_map_point_x,
														dev_best_stroke_map_point_y, dev_best_stroke_map_R, dev_best_stroke_map_G, 
														dev_best_stroke_map_B, dev_grad_hx, dev_grad_hy, dev_M, dev_u, dev_new_u,
														dev_v, dev_new_v, dev_p, dev_gR, dev_gG, dev_gB, dev_dR, dev_dG, dev_dB,
														dev_new_gR, dev_new_gG, dev_new_gB, dev_new_dR, dev_new_dG, dev_new_dB,
														dev_gauss_filter, dev_gauss_M, dev_h, dev_best_x, dev_best_y, in->width, in->height, t);
			
			hipDeviceSynchronize();
			//printCudaLastError();

			//描画したストロークの周囲のみ改善値マップをリセット
			gpu_reset_improved_value_map<<<1,1>>>(dev_GLOBAL_improved_value_map, dev_best_stroke_map_pnum, dev_best_stroke_map_point_x, dev_best_stroke_map_point_y, dev_best_x, dev_best_y, in->width, in->height, t);
			hipDeviceSynchronize();
			//printCudaLastError();

			//hipEventRecord(end, 0);//計測ストップ

			//実行時間表示
			//hipEventElapsedTime(&ftimer, start, end);
			//printf("%f,ms\n", ftimer);

			//キャンバスをGPUからCPUにコピー
			hipMemcpy(nimgR_1dim, dev_nimgR, in->width*in->height*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(nimgG_1dim, dev_nimgG, in->width*in->height*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(nimgB_1dim, dev_nimgB, in->width*in->height*sizeof(int), hipMemcpyDeviceToHost);
			
			//1次元配列を2次元配列に変換
			nimgR->data = ReshapeInt_1to2(nimgR_1dim, in->width, in->height);
			nimgG->data = ReshapeInt_1to2(nimgG_1dim, in->width, in->height);
			nimgB->data = ReshapeInt_1to2(nimgB_1dim, in->width, in->height);
			nimgC->dataR = nimgR->data;
			nimgC->dataG = nimgG->data;
			nimgC->dataB = nimgB->data;
			
			//一定ストロークごとに途中経過画像を書き出す
			paint_count++;
			nc++;
			if(nc%100==0 || nc<=100){
				strcpy(out_filename, dir_path);
				strcat(out_filename, in_filename);
				snprintf(count_name, 16, "%d", nc);
				strcat(out_filename, "_s");
				strcat(out_filename, count_name);
				strcat(out_filename, ".ppm");
				write_ppm(out_filename, nimgC);
				//printf("%s\n",out_filename);
				//printf("%d:",t);
				//pd("TIME[s]",my_clock());
			}
		}

		//現在の半径の完成画像を出力
		strcpy(out_filename, dir_path);
		strcat(out_filename, in_filename);
		snprintf(count_name, 16, "%02d", t);
		strcat(out_filename, "__t");
		strcat(out_filename, count_name);
		strcat(out_filename, ".ppm");
		write_ppm(out_filename, nimgC);
		//printf("%s\n",out_filename);
		//printf("%d:",t);
		//pd("TIME[s]",my_clock());

		printf("\n////////////////////\nt%d done.\n////////////////////\n\n\n",t);
		p("Paint_num",paint_count); paint_count=0;
		paint_count=0;
	}

	//第一段階描画後の中間画像を出力
	snprintf(count_name, 16, "%03d", tc);
	strcpy(out_filename, dir_path);
	strcat(out_filename, in_filename);
	strcat(out_filename, "__t");
	strcat(out_filename, count_name);
	strcat(out_filename, ".ppm");
	write_ppm(out_filename, nimgC);
	printf("%s\n",out_filename);
	tc=t;
	printf("%d",t);
	pd("TIME[s]",my_clock());

	//---------------------------
	//エッジマップを計算し、エッジの複雑な周辺だけに描画を行う
	//---------------------------
	int lc=0;
	loop_cont = opt2_loop_cont;
	min_stroke = opt2_min_stroke;
	ratio = opt2_ratio;
	tc=-1;
	double maxValue=0.30, minValue=0.10;
	PGM *canny;
	PGM *EdgeMap;

	thick_max = opt2_thick_max;
	if(thick_max){
		strcat(log_sentence, "\r\n\r\n[EdgeStroke]\r\n");
		canny = cannyedge_detector(gray, maxValue, minValue, thick_min);
		EdgeMap = calcu_EdgeMap(canny, thick_min, sobel_angle);
		//EdgeMap = expand_Edge(canny, thick_min);
	}
	thick_min = opt2_thick_min;

	Add_dictionary_to_sentence_d(log_sentence, "CannyTIME[s]", my_clock());
	pd("Canny:TIME[s]",my_clock());

	for(t=thick_max; t>=thick_min; t--){
		paint_count=0;

		//ストロークサイズのガウスフィルタを生成
		gauce_filter = create_dally(2*t+1, 2*t+1);
		sigma = t/3.0;
		for(i=0; i<2*t+1; i++){
			for(j=0; j<2*t+1; j++){
				gauce_filter[i][j] = gause_func(i-t, j-t, sigma);
			}
		}

		for(y=y_defo; y<in->height; y=y+t) {  //ウィンドウの大きさに合わせて
			for(x=x_defo; x<in->width; x=x+t) {  //ウィンドウをずらす距離を変えとく
				//エッジマップにない箇所なら次のループに
				if(EdgeMap->data[x][y]!=255) continue;

				//ウィンドウの中の差分の合計を取る
				diff_sum = break_flag = pnum = 0;

				offscrn_count = 0;
				for(xc=-t; xc<=t; xc++) {
					if((x+xc)<0 || (x+xc)>in->width-1) {offscrn_count += 2*t+1;		continue;	}
					for(yc=-t; yc<=t; yc++) {
						if((y+yc)<0 || (y+yc)>in->height-1) {	offscrn_count++;
						}else{
							diff_sum += abs(nimgR->data[x+xc][y+yc] - cmprR->data[x+xc][y+yc]);
							diff_sum += abs(nimgG->data[x+xc][y+yc] - cmprG->data[x+xc][y+yc]);
							diff_sum += abs(nimgB->data[x+xc][y+yc] - cmprB->data[x+xc][y+yc]);
						}
					}
				}
				diff_sum = diff_sum/((2*t+1)*(2*t+1)-offscrn_count)/3;

				//差分の合計平均(画素当たりの差分)が一定以上ならストローク開始位置とする
				if(diff_sum < window_diff_border) {
					stroke_histogram[pnum]++;
					continue;
				}
				pnum=1;		//第一点確定
				p[0].x=x+0.5; p[0].y=y+0.5;

				//一つ目の描画領域から描画色を平均を取って取得
				if(opt_USE_calcu_color_bi){
					bright.R = calcu_color_bi(cmprR->data, cmprR->width, cmprR->height, x, y, t, 50, gauce_filter);
					bright.G = calcu_color_bi(cmprG->data, cmprG->width, cmprG->height, x, y, t, 50, gauce_filter);
					bright.B = calcu_color_bi(cmprB->data, cmprB->width, cmprB->height, x, y, t, 50, gauce_filter);
				} else{
					bright.R = calcu_color(cmprR->data, cmprR->width, cmprR->height, x, y, t);
					bright.G = calcu_color(cmprG->data, cmprG->width, cmprG->height, x, y, t);
					bright.B = calcu_color(cmprB->data, cmprB->width, cmprB->height, x, y, t);
				}


				theta =  calcu_histogram(cmpr, sobel_abs, sobel_angle, histogram_partition,
						gauce_filter, p[0].x, p[0].y, t, &break_flag);


				//制御点を方向から計算し代入
				p[1] = calcu_point(cmpr, p[0], t, theta);


				//二つ目の描画点周りの色が描画色と一致するか確認する
				if(opt_USE_Lab_ColorDiff){
					sum = diffsum_Lab(in_Lab, nimgC, p[1], t, bright, 1.0);
				} else{
					sum = 0;
					sum += diffsum_clr(cmprR, nimgR, p[1], t, bright.R);
					sum += diffsum_clr(cmprG, nimgG, p[1], t, bright.G);
					sum += diffsum_clr(cmprB, nimgB, p[1], t, bright.B);
				}

				//二つ目の制御点周りの色が描画色としきい値以上の差を持つなら描画せず反対方向の制御点を見る
				if( sum < color_diff_border){
					//もう一つの勾配垂直の点を代入
					theta += PI;
					p[1] = calcu_point(cmpr, p[0], t, theta);

					//反対方向の第二点の描画点周りの色が描画色と一致するか確認する
					if(opt_USE_Lab_ColorDiff){
						sum = diffsum_Lab(in_Lab, nimgC, p[1], t, bright, 1.0);
					} else{
						sum = 0;
						sum += diffsum_clr(cmprR, nimgR, p[1], t, bright.R);
						sum += diffsum_clr(cmprG, nimgG, p[1], t, bright.G);
						sum += diffsum_clr(cmprB, nimgB, p[1], t, bright.B);
					}

					//どちらの第二点も不適切なら描画をせず次のループへ
					if( sum < color_diff_border) {
						continue;
					}
				}
				//適切な第二点が見つかれば次へ
				pnum=2;		//第二点確定


				/*
					POINT2から勾配により次の制御点を探していく
				*/

				while(pnum<max_stroke){
					former_theta=theta;

					//第pnum点周りにおいて、sobelからヒストグラムを作成し最大のものを勾配とする
					theta =  calcu_histogram(cmpr, sobel_abs, sobel_angle, histogram_partition,
											gauce_filter, p[pnum-1].x, p[pnum-1].y, t, &break_flag);

					//制御点の為す角が急峻になるようなら逆方向に角度を取る
					if( (theta < former_theta-PI/2) || (theta > former_theta+PI/2) ) {theta += PI;}
					p[pnum] = calcu_point(cmpr, p[pnum-1], t, theta);

					//pnum+1目の描画点周りの色が描画色と一致するか確認する  //点当たりの差異平均
					if(opt_USE_Lab_ColorDiff){
						sum = diffsum_Lab(in_Lab, nimgC, p[pnum], t, bright, 1.0);
					} else{
						sum = 0;
						sum += diffsum_clr(cmprR, nimgR, p[pnum], t, bright.R);
						sum += diffsum_clr(cmprG, nimgG, p[pnum], t, bright.G);
						sum += diffsum_clr(cmprB, nimgB, p[pnum], t, bright.B);
					}

					/*
						pnum+1目の(次の)制御点周りの色が描画色としきい値以上の差を持つなら
						それまでの制御点を用いて線を描画
					*/
					if( sum < color_diff_border) {break_flag=1; break;}
					else {pnum++;}

				}

				if(pnum>=min_stroke) {
					//算出したpnum個の制御点を用いてストロークを描画
					Paint_Bezier_ex(p, pnum, nimgR, t, bright.R, ratio);
					Paint_Bezier_ex(p, pnum, nimgG, t, bright.G, ratio);
					Paint_Bezier_ex(p, pnum, nimgB, t, bright.B, ratio);

				}

				paint_count++;
				nc++;
			}
		}


		printf("////////////////////\nt%d done.\n////////////////////\n\n\n",t);
		Free_dally(gauce_filter, 2*t+1);


		{
			tc=t;
			strcpy(out_filename, dir_path);
			strcat(out_filename, in_filename);
			snprintf(count_name, 16, "%02d", t);
			strcat(out_filename, "__st");
			strcat(out_filename, count_name);
			snprintf(count_name, 16, "%02d", lc);
			strcat(out_filename, "_lc");
			strcat(out_filename, count_name);
			strcat(out_filename, ".ppm");
			write_ppm(out_filename, nimgC);
			printf("%s\n",out_filename);
			printf("%d:",t);
			strcat(log_sentence, "\r\n");
			Add_dictionary_to_sentence(log_sentence, "t", t);
			Add_dictionary_to_sentence(log_sentence, "s_count", paint_count);
			Add_dictionary_to_sentence_d(log_sentence, "TIME[s]", my_clock());
			pd("TIME[s]",my_clock());
		}

		lc++;		//同じ半径でのループをcont回する
		if((lc%loop_cont) != 0){
			p("lc",lc);
			t++;
			x_defo += t/loop_cont;
			y_defo += t/loop_cont;
			if(t/loop_cont==0){
				x_defo++;	y_defo++;
			}
		}
	}

	if(thick_max){
		strcpy(out_filename, dir_path);
		strcat(out_filename, in_filename);
		strcat(out_filename, "__EdgeMap");
		strcat(out_filename, ".pgm");
		if(write_pgm(out_filename, EdgeMap)){ printf("WRITE PNG ERROR.");}
	}

	double MSE = image_MSE(nimgC, in);
	Add_dictionary_to_sentence(log_sentence, "MSE", (int)MSE);

	Add_dictionary_to_sentence_d(log_sentence, "All_Execution_TIME", my_clock());
	printf("%s\n", log_filename);
	if(log_print(log_filename, log_sentence, "w") ){ printf("LOG_PRINTING_FAIL\n"); }


	Free_dally(sobel_abs, in->width);
	Free_dally(sobel_angle, in->width);
	Free_dally(h, in->width);
	Free_dally(grad_hx, in->width+1);
	Free_dally(grad_hy, in->width);
	FreePGM(gray);
	FreePGM(inR);
	FreePGM(inG);
	FreePGM(inB);
	FreePGM(nimgV);
	free(nimgR);
	free(nimgG);
	free(nimgB);
	if(thick_max){
		FreePGM(canny);
		FreePGM(EdgeMap);
	}

	hipFree(dev_GLOBAL_improved_value_map);
	hipFree(dev_PerlinNoise);
	hipFree(dev_in_Lab_L);
	hipFree(dev_in_Lab_a);
	hipFree(dev_in_Lab_b);
	hipFree(dev_cmpr);
	hipFree(dev_cmprR);
	hipFree(dev_cmprG);
	hipFree(dev_cmprB);
	hipFree(dev_nimgR);
	hipFree(dev_nimgG);
	hipFree(dev_nimgB);
	hipFree(dev_best_stroke_map_pnum);
	hipFree(dev_best_stroke_map_point_x);
	hipFree(dev_best_stroke_map_point_y);
	hipFree(dev_best_stroke_map_R);
	hipFree(dev_best_stroke_map_G);
	hipFree(dev_best_stroke_map_B);
	hipFree(dev_sobel_abs);
	hipFree(dev_sobel_angle);
	hipFree(dev_grad_hx);
	hipFree(dev_grad_hy);
	hipFree(dev_M);
	hipFree(dev_u);
	hipFree(dev_new_u);
	hipFree(dev_v);
	hipFree(dev_new_v);
	hipFree(dev_p);
	hipFree(dev_gR);
	hipFree(dev_gG);
	hipFree(dev_gB);
	hipFree(dev_dR);
	hipFree(dev_dG);
	hipFree(dev_dB);
	hipFree(dev_gauss_M);
	hipFree(dev_h);

    return nimgC;
}