#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/stat.h>
#include <dirent.h>
//#include <png.h>
//#include <jpeglib.h>
#include <omp.h>
#include "ImageIO/image.h"
#include "sbr.h"
#include "sbr_opt.h"
#include "water.h"

void printCudaLastError(){
	hipError_t err = hipGetLastError();
	printf("hipGetLastError::%s(code:%d)\n",hipGetErrorString(err),err);
	if(err)	exit(0);
}

__global__ void test(){
	__shared__ float s_diff_sum[1];
    __shared__ float s_diff_R[1];
    __shared__ float s_diff_G[1];
    __shared__ float s_diff_B[1];
	__shared__ int s_offscrn_count[1];
    __shared__ float s_theta[1];
    __shared__ float s_histogram[opt_histogram_partition];
    __shared__ float s_error_sum[1];
    __shared__ char s_test_Canvas_R[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ char s_test_Canvas_G[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ char s_test_Canvas_B[(opt_thick_max*(opt_max_stroke+2))*(opt_thick_max*(opt_max_stroke+2))];
    __shared__ float s_max_velocity[1];
    __shared__ float s_delta_MAX[1];

	for(int y=blockIdx.y; y<1; y=y+gridDim.y) {
		for(int x=blockIdx.x; x<1; x=x+gridDim.x) {

			if((threadIdx.x==0)&&(threadIdx.y==0)){
				s_diff_sum[0] = 0;
				s_diff_R[0] = 0;
				s_diff_G[0] = 0;
				s_diff_B[0] = 0;
				s_offscrn_count[0] = 0;
			}
			__syncthreads();

			if((threadIdx.x==0)&&(threadIdx.y==0)){
				printf("s_offscrn_count = %d\n",s_offscrn_count[0]);
				printf("s_diff_R = %f\n",s_diff_R[0]);
				printf("s_diff_G = %f\n",s_diff_G[0]);
				printf("s_diff_B = %f\n",s_diff_B[0]);
				printf("s_diff_sum = %f\n",s_diff_sum[0]);
			}
		}
	}
}

int main(){

	dim3 block_num(1, 1);
	dim3 thread_num(32, 32);

	test<<<block_num,thread_num>>>();
	hipDeviceSynchronize();
	printCudaLastError();
}